
#include<stdio.h>
#include<gq58x4.h>

#define NN       100000000UL

int main(void){ 
   long i; unsigned int sum=0;
   gq58x4_state state;
   gq58x4_init_(&state);
   gq58x4_print_state_(&state);
   for(i=0;i<NN;i++) sum+=gq58x4_generate_(&state);
   printf("%ld GQ58X4 pseudorandom numbers generated using general instructions of CPU.\n",NN);
   printf("Fractional part of the total sum of generated numbers: %f\n",sum/4294967296.);
   printf("Next output value: %f\n",gq58x4_generate_(&state)/4294967296.);
   return 0;
}
