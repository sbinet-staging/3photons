
#include<stdio.h>
#include<gm61.h>

#define NN       100000000UL

int main(void){ 
   long i; unsigned int sum=0;
   gm61_state state;
   gm61_init_(&state);
   gm61_print_state_(&state);
   for(i=0;i<NN;i++) sum+=gm61_generate_(&state);
   printf("%ld GM61 pseudorandom numbers generated using general instructions of CPU.\n",NN);
   printf("Fractional part of the total sum of generated numbers: %f\n",sum/4294967296.);
   printf("Next output value: %f\n",gm61_generate_(&state)/4294967296.);
   return 0;
}
