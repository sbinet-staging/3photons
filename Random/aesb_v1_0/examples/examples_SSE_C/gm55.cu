
#include<stdio.h>
#include<gm55.h>

#define NN       100000000UL

int main(void){ 
   long i; unsigned int sum=0;
   gm55_state state; gm55_sse_state sse_state;
   gm55_init_(&state); gm55_get_sse_state_(&state,&sse_state);
   gm55_print_state_(&state);
   for(i=0;i<NN;i++) sum+=gm55_sse_generate_(&sse_state);
   printf("%ld GM55 pseudorandom numbers generated using SSE instructions and 128-bit XMM-registers of CPU.\n",NN);
   printf("Fractional part of the total sum of generated numbers: %f\n",sum/4294967296.);
   printf("Next output value: %f\n",gm55_sse_generate_(&sse_state)/4294967296.);
   return 0;
}
