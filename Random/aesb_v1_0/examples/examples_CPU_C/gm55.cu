
#include<stdio.h>
#include<gm55.h>

#define NN       100000000UL

int main(void){ 
   long i; unsigned int sum=0;
   gm55_state state;
   gm55_init_(&state);
   gm55_print_state_(&state);
   for(i=0;i<NN;i++) sum+=gm55_generate_(&state);
   printf("%ld GM55 pseudorandom numbers generated using general instructions of CPU.\n",NN);
   printf("Fractional part of the total sum of generated numbers: %f\n",sum/4294967296.);
   printf("Next output value: %f\n",gm55_generate_(&state)/4294967296.);
   return 0;
}
