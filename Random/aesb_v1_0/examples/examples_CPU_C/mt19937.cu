
#include<stdio.h>
#include<mt19937.h>

#define NN       100000000UL

int main(void){ 
   long i; unsigned int sum=0;
   mt19937_state state;
   mt19937_init_(&state);
   mt19937_print_state_(&state);
   for(i=0;i<NN;i++) sum+=mt19937_generate_(&state);
   printf("%ld MT19937 pseudorandom numbers generated using general instructions of CPU.\n",NN);
   printf("Fractional part of the total sum of generated numbers: %f\n",sum/4294967296.);
   printf("Next output value: %f\n",mt19937_generate_(&state)/4294967296.);
   return 0;
}
