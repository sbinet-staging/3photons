// (c) Copyright 2013 Lev Barash, Landau Institute for Theoretical Physics, Russian Academy of Sciences
// This is supplement to the paper:
// L.Yu. Barash, L.N. Shchur, "PRAND: GPU accelerated parallel random number generation library: Using most reliable algorithms and applying parallelism of modern GPUs and CPUs".
// e-mail: barash @ itp.ac.ru (remove space)


#include <hip/hip_runtime.h>
#include<stdio.h>

#define gm31_CUDA_CALL(x) do { if((x) != hipSuccess) { printf("Error: %s at %s:%d\n",hipGetErrorString(hipGetLastError()),__FILE__,__LINE__); exit(1);}} while(0)

#define gm31_BLOCKS  512
#define gm31_THREADS 128
#define gm31_ARRAY_SECTIONS  (gm31_BLOCKS*gm31_THREADS/32)

#define gm31_qg    30064771058ULL
#define gm31_g     2147483647
#define gm31_halfg 1073741824
#define gm31_k     11
#define gm31_q     14

typedef struct{
  unsigned xN[32],xP[32];
} gm31_state;

typedef struct{
  unsigned xN[64] __attribute__ ((aligned(16))),
           xP[64] __attribute__ ((aligned(16)));
} gm31_sse_state;

unsigned gm31_Consts[16] __attribute__ ((aligned(16))) =
{4294967222UL,36UL,4294967222UL,36UL,gm31_k,0,gm31_k,0,gm31_q,0,gm31_q,0,gm31_g,0,gm31_g,0};

extern "C" __host__ unsigned int gm31_sse_generate_(gm31_sse_state* state){
  unsigned output1,output2;
  asm volatile("\n" \
      "movaps 48(%3),%%xmm5\n" \
      "\n" \
      "movaps (%1),%%xmm0\n" \
      "movaps %%xmm0,%%xmm7\n" \
      "movaps (%2),%%xmm6\n" \
      "pmuludq 16(%3),%%xmm0\n" \
      "pmuludq 32(%3),%%xmm6\n" \
      "paddq (%3),%%xmm0\n" \
      "psubq %%xmm6,%%xmm0\n" \
      "movaps %%xmm0,%%xmm6\n" \
      "psrlq $31,%%xmm6\n" \
      "andps %%xmm5,%%xmm0\n" \
      "paddq %%xmm6,%%xmm0\n" \
      "movaps %%xmm0,(%1)\n" \
      "movaps %%xmm7,(%2)\n" \
      "\n" \
      "movaps 16(%1),%%xmm1\n" \
      "movaps %%xmm1,%%xmm7\n" \
      "movaps 16(%2),%%xmm6\n" \
      "pmuludq 16(%3),%%xmm1\n" \
      "pmuludq 32(%3),%%xmm6\n" \
      "paddq (%3),%%xmm1\n" \
      "psubq %%xmm6,%%xmm1\n" \
      "movaps %%xmm1,%%xmm6\n" \
      "psrlq $31,%%xmm6\n" \
      "andps %%xmm5,%%xmm1\n" \
      "paddq %%xmm6,%%xmm1\n" \
      "movaps %%xmm1,16(%1)\n" \
      "movaps %%xmm7,16(%2)\n" \
      "\n" \
      "movaps 32(%1),%%xmm2\n" \
      "movaps %%xmm2,%%xmm7\n" \
      "movaps 32(%2),%%xmm6\n" \
      "pmuludq 16(%3),%%xmm2\n" \
      "pmuludq 32(%3),%%xmm6\n" \
      "paddq (%3),%%xmm2\n" \
      "psubq %%xmm6,%%xmm2\n" \
      "movaps %%xmm2,%%xmm6\n" \
      "psrlq $31,%%xmm6\n" \
      "andps %%xmm5,%%xmm2\n" \
      "paddq %%xmm6,%%xmm2\n" \
      "movaps %%xmm2,32(%1)\n" \
      "movaps %%xmm7,32(%2)\n" \
      "\n" \
      "movaps 48(%1),%%xmm3\n" \
      "movaps %%xmm3,%%xmm7\n" \
      "movaps 48(%2),%%xmm6\n" \
      "pmuludq 16(%3),%%xmm3\n" \
      "pmuludq 32(%3),%%xmm6\n" \
      "paddq (%3),%%xmm3\n" \
      "psubq %%xmm6,%%xmm3\n" \
      "movaps %%xmm3,%%xmm6\n" \
      "psrlq $31,%%xmm6\n" \
      "andps %%xmm5,%%xmm3\n" \
      "paddq %%xmm6,%%xmm3\n" \
      "movaps %%xmm3,48(%1)\n" \
      "movaps %%xmm7,48(%2)\n" \
      "\n" \
      "shufps $136,%%xmm1,%%xmm0\n" \
      "shufps $136,%%xmm3,%%xmm2\n" \
      "psrld  $30,%%xmm0\n" \
      "psrld  $30,%%xmm2\n" \
      "packssdw %%xmm2,%%xmm0\n" \
      "\n" \
      "movaps 64(%1),%%xmm1\n" \
      "movaps %%xmm1,%%xmm7\n" \
      "movaps 64(%2),%%xmm6\n" \
      "pmuludq 16(%3),%%xmm1\n" \
      "pmuludq 32(%3),%%xmm6\n" \
      "paddq (%3),%%xmm1\n" \
      "psubq %%xmm6,%%xmm1\n" \
      "movaps %%xmm1,%%xmm6\n" \
      "psrlq $31,%%xmm6\n" \
      "andps %%xmm5,%%xmm1\n" \
      "paddq %%xmm6,%%xmm1\n" \
      "movaps %%xmm1,64(%1)\n" \
      "movaps %%xmm7,64(%2)\n" \
      "\n" \
      "movaps 80(%1),%%xmm2\n" \
      "movaps %%xmm2,%%xmm7\n" \
      "movaps 80(%2),%%xmm6\n" \
      "pmuludq 16(%3),%%xmm2\n" \
      "pmuludq 32(%3),%%xmm6\n" \
      "paddq (%3),%%xmm2\n" \
      "psubq %%xmm6,%%xmm2\n" \
      "movaps %%xmm2,%%xmm6\n" \
      "psrlq $31,%%xmm6\n" \
      "andps %%xmm5,%%xmm2\n" \
      "paddq %%xmm6,%%xmm2\n" \
      "movaps %%xmm2,80(%1)\n" \
      "movaps %%xmm7,80(%2)\n" \
      "\n" \
      "movaps 96(%1),%%xmm3\n" \
      "movaps %%xmm3,%%xmm7\n" \
      "movaps 96(%2),%%xmm6\n" \
      "pmuludq 16(%3),%%xmm3\n" \
      "pmuludq 32(%3),%%xmm6\n" \
      "paddq (%3),%%xmm3\n" \
      "psubq %%xmm6,%%xmm3\n" \
      "movaps %%xmm3,%%xmm6\n" \
      "psrlq $31,%%xmm6\n" \
      "andps %%xmm5,%%xmm3\n" \
      "paddq %%xmm6,%%xmm3\n" \
      "movaps %%xmm3,96(%1)\n" \
      "movaps %%xmm7,96(%2)\n" \
      "\n" \
      "movaps 112(%1),%%xmm4\n" \
      "movaps %%xmm4,%%xmm7\n" \
      "movaps 112(%2),%%xmm6\n" \
      "pmuludq 16(%3),%%xmm4\n" \
      "pmuludq 32(%3),%%xmm6\n" \
      "paddq (%3),%%xmm4\n" \
      "psubq %%xmm6,%%xmm4\n" \
      "movaps %%xmm4,%%xmm6\n" \
      "psrlq $31,%%xmm6\n" \
      "andps %%xmm5,%%xmm4\n" \
      "paddq %%xmm6,%%xmm4\n" \
      "movaps %%xmm4,112(%1)\n" \
      "movaps %%xmm7,112(%2)\n" \
      "\n" \
      "shufps $136,%%xmm2,%%xmm1\n" \
      "shufps $136,%%xmm4,%%xmm3\n" \
      "psrld  $30,%%xmm1\n" \
      "psrld  $30,%%xmm3\n" \
      "packssdw %%xmm3,%%xmm1\n" \
      "\n" \
      "packsswb %%xmm1,%%xmm0\n" \
      "psllw $7,%%xmm0\n" \
      "pmovmskb %%xmm0,%0\n" \
      "":"=r"(output1):"r"(state->xN),"r"(state->xP),"r"(gm31_Consts));
  asm volatile("\n" \
      "movaps 128(%1),%%xmm0\n" \
      "movaps %%xmm0,%%xmm7\n" \
      "movaps 128(%2),%%xmm6\n" \
      "pmuludq 16(%3),%%xmm0\n" \
      "pmuludq 32(%3),%%xmm6\n" \
      "paddq (%3),%%xmm0\n" \
      "psubq %%xmm6,%%xmm0\n" \
      "movaps %%xmm0,%%xmm6\n" \
      "psrlq $31,%%xmm6\n" \
      "andps %%xmm5,%%xmm0\n" \
      "paddq %%xmm6,%%xmm0\n" \
      "movaps %%xmm0,128(%1)\n" \
      "movaps %%xmm7,128(%2)\n" \
      "\n" \
      "movaps 144(%1),%%xmm1\n" \
      "movaps %%xmm1,%%xmm7\n" \
      "movaps 144(%2),%%xmm6\n" \
      "pmuludq 16(%3),%%xmm1\n" \
      "pmuludq 32(%3),%%xmm6\n" \
      "paddq (%3),%%xmm1\n" \
      "psubq %%xmm6,%%xmm1\n" \
      "movaps %%xmm1,%%xmm6\n" \
      "psrlq $31,%%xmm6\n" \
      "andps %%xmm5,%%xmm1\n" \
      "paddq %%xmm6,%%xmm1\n" \
      "movaps %%xmm1,144(%1)\n" \
      "movaps %%xmm7,144(%2)\n" \
      "\n" \
      "movaps 160(%1),%%xmm2\n" \
      "movaps %%xmm2,%%xmm7\n" \
      "movaps 160(%2),%%xmm6\n" \
      "pmuludq 16(%3),%%xmm2\n" \
      "pmuludq 32(%3),%%xmm6\n" \
      "paddq (%3),%%xmm2\n" \
      "psubq %%xmm6,%%xmm2\n" \
      "movaps %%xmm2,%%xmm6\n" \
      "psrlq $31,%%xmm6\n" \
      "andps %%xmm5,%%xmm2\n" \
      "paddq %%xmm6,%%xmm2\n" \
      "movaps %%xmm2,160(%1)\n" \
      "movaps %%xmm7,160(%2)\n" \
      "\n" \
      "movaps 176(%1),%%xmm3\n" \
      "movaps %%xmm3,%%xmm7\n" \
      "movaps 176(%2),%%xmm6\n" \
      "pmuludq 16(%3),%%xmm3\n" \
      "pmuludq 32(%3),%%xmm6\n" \
      "paddq (%3),%%xmm3\n" \
      "psubq %%xmm6,%%xmm3\n" \
      "movaps %%xmm3,%%xmm6\n" \
      "psrlq $31,%%xmm6\n" \
      "andps %%xmm5,%%xmm3\n" \
      "paddq %%xmm6,%%xmm3\n" \
      "movaps %%xmm3,176(%1)\n" \
      "movaps %%xmm7,176(%2)\n" \
      "\n" \
      "shufps $136,%%xmm1,%%xmm0\n" \
      "shufps $136,%%xmm3,%%xmm2\n" \
      "psrld  $30,%%xmm0\n" \
      "psrld  $30,%%xmm2\n" \
      "packssdw %%xmm2,%%xmm0\n" \
      "\n" \
      "movaps 192(%1),%%xmm1\n" \
      "movaps %%xmm1,%%xmm7\n" \
      "movaps 192(%2),%%xmm6\n" \
      "pmuludq 16(%3),%%xmm1\n" \
      "pmuludq 32(%3),%%xmm6\n" \
      "paddq (%3),%%xmm1\n" \
      "psubq %%xmm6,%%xmm1\n" \
      "movaps %%xmm1,%%xmm6\n" \
      "psrlq $31,%%xmm6\n" \
      "andps %%xmm5,%%xmm1\n" \
      "paddq %%xmm6,%%xmm1\n" \
      "movaps %%xmm1,192(%1)\n" \
      "movaps %%xmm7,192(%2)\n" \
      "\n" \
      "movaps 208(%1),%%xmm2\n" \
      "movaps %%xmm2,%%xmm7\n" \
      "movaps 208(%2),%%xmm6\n" \
      "pmuludq 16(%3),%%xmm2\n" \
      "pmuludq 32(%3),%%xmm6\n" \
      "paddq (%3),%%xmm2\n" \
      "psubq %%xmm6,%%xmm2\n" \
      "movaps %%xmm2,%%xmm6\n" \
      "psrlq $31,%%xmm6\n" \
      "andps %%xmm5,%%xmm2\n" \
      "paddq %%xmm6,%%xmm2\n" \
      "movaps %%xmm2,208(%1)\n" \
      "movaps %%xmm7,208(%2)\n" \
      "\n" \
      "movaps 224(%1),%%xmm3\n" \
      "movaps %%xmm3,%%xmm7\n" \
      "movaps 224(%2),%%xmm6\n" \
      "pmuludq 16(%3),%%xmm3\n" \
      "pmuludq 32(%3),%%xmm6\n" \
      "paddq (%3),%%xmm3\n" \
      "psubq %%xmm6,%%xmm3\n" \
      "movaps %%xmm3,%%xmm6\n" \
      "psrlq $31,%%xmm6\n" \
      "andps %%xmm5,%%xmm3\n" \
      "paddq %%xmm6,%%xmm3\n" \
      "movaps %%xmm3,224(%1)\n" \
      "movaps %%xmm7,224(%2)\n" \
      "\n" \
      "movaps 240(%1),%%xmm4\n" \
      "movaps %%xmm4,%%xmm7\n" \
      "movaps 240(%2),%%xmm6\n" \
      "pmuludq 16(%3),%%xmm4\n" \
      "pmuludq 32(%3),%%xmm6\n" \
      "paddq (%3),%%xmm4\n" \
      "psubq %%xmm6,%%xmm4\n" \
      "movaps %%xmm4,%%xmm6\n" \
      "psrlq $31,%%xmm6\n" \
      "andps %%xmm5,%%xmm4\n" \
      "paddq %%xmm6,%%xmm4\n" \
      "movaps %%xmm4,240(%1)\n" \
      "movaps %%xmm7,240(%2)\n" \
      "\n" \
      "shufps $136,%%xmm2,%%xmm1\n" \
      "shufps $136,%%xmm4,%%xmm3\n" \
      "psrld  $30,%%xmm1\n" \
      "psrld  $30,%%xmm3\n" \
      "packssdw %%xmm3,%%xmm1\n" \
      "\n" \
      "packsswb %%xmm1,%%xmm0\n" \
      "psllw $7,%%xmm0\n" \
      "pmovmskb %%xmm0,%0\n" \
      "shll $16,%0\n" \
      "":"=r"(output2):"r"(state->xN),"r"(state->xP),"r"(gm31_Consts));
  asm volatile("\n" \
      "addl %1,%0\n" \
      "\n" \
      "":"=r"(output2):"r"(output1),"0"(output2));
  return output2;
}

extern "C" __device__ __host__ void gm31_get_sse_state_(gm31_state* state,gm31_sse_state* sse_state){
  int i;
  for(i=0;i<32;i++) {
    sse_state->xN[2*i]=state->xN[i]; sse_state->xP[2*i]=state->xP[i];
    sse_state->xN[2*i+1]=sse_state->xP[2*i+1]=0;
  }
}

extern "C" __device__ __host__ unsigned gm31_mod_g(unsigned long long x){ // returns x (mod g)
  unsigned long long F,G; G=x;
  do{ F=(G>>31); G = (G-(F<<31)+F); } while(G>gm31_g);
  return G;
}

extern "C" __device__ __host__ unsigned gm31_CNext(unsigned N,unsigned P){
  unsigned long long curr1,curr2,curr3;
  curr1=(unsigned long long)gm31_k*(unsigned long long)N; 
  curr2=(unsigned long long)gm31_q*(unsigned long long)P; 
  curr3=gm31_mod_g(gm31_qg+curr1-curr2); return curr3;
}

extern "C" __device__ __host__ unsigned gm31_CNext2(unsigned N,unsigned P,unsigned myk,unsigned myq){
  unsigned long long curr1,curr2,curr3;
  curr1=(unsigned long long)myk*(unsigned long long)N;
  curr2=(unsigned long long)myq*(unsigned long long)P;
  curr3=gm31_mod_g((unsigned long long)myq*(unsigned long long)gm31_g+curr1-curr2);
  return curr3;
}

extern "C" __device__ __host__ unsigned gm31_GetNextN(unsigned x0,unsigned x1,unsigned n){ // returns x_{2^n}
  unsigned myk=gm31_k,myq=gm31_q,i,x=x1;
  for(i=0;i<n;i++){
    x=gm31_CNext2(x,x0,myk,myq);
    myk=gm31_CNext2(myk,2,myk,myq);
    myq=gm31_CNext2(myq,0,myq,0);
  }
  return x;
}

extern "C" __device__ __host__ unsigned gm31_GetNextAny(unsigned x0,unsigned x1,unsigned long long N){ // returns x_N
  unsigned long long i; unsigned xp=x0,xn=x1,xpnew,xnnew,shift=0;
  i=N; while(i>0){
    if(i%2==1){                        // xp,xn ----> 2^shift
      xpnew=gm31_GetNextN(xp,xn,shift);
      xnnew=gm31_GetNextN(xn,gm31_CNext(xn,xp),shift);
      xp=xpnew; xn=xnnew;
    }
    i/=2; shift++;
  }
  return xp;
}

extern "C" __device__ __host__ void gm31_skipahead_(gm31_state* state, unsigned long long offset){
  unsigned xn,xp,j; 
  for(j=0;j<32;j++){
    xp=gm31_GetNextAny(state->xP[j],state->xN[j],offset);
    xn=gm31_GetNextAny(state->xP[j],state->xN[j],offset+1);
    state->xP[j]=xp; state->xN[j]=xn;
  }
}

extern "C" __device__ __host__ void gm31_init_(gm31_state* state){
  unsigned x0=554937932UL,x1=1253942293UL,xp,xn,j;
  for(j=0;j<32;j++){
    xp=gm31_GetNextAny(x0,x1,144115183781032008ULL);
    xn=gm31_GetNextAny(x0,x1,144115183781032009ULL);
    state->xP[j]=xp; state->xN[j]=xn; x0=xp; x1=xn;
  }
}

extern "C" __device__ __host__ void gm31_init_short_sequence_(gm31_state* state,unsigned SequenceNumber){
  gm31_init_(state);                     // 0 <= SequenceNumber < 10^9;   length of each sequence <= 8*10^7
  gm31_skipahead_(state,82927047ULL*(unsigned long long)SequenceNumber);
}

extern "C" __device__ __host__ void gm31_init_medium_sequence_(gm31_state* state,unsigned SequenceNumber){
  gm31_init_(state);                     // 0 <= SequenceNumber < 10^7;   length of each sequence <= 8*10^9
  gm31_skipahead_(state,8799201913ULL*(unsigned long long)SequenceNumber);
}

extern "C" __device__ __host__ void gm31_init_long_sequence_(gm31_state* state,unsigned SequenceNumber){
  gm31_init_(state);                     // 0 <= SequenceNumber < 10^5;   length of each sequence <= 8*10^11
  gm31_skipahead_(state,828317697521ULL*(unsigned long long)SequenceNumber);
}

extern "C" __device__ __host__ unsigned int gm31_generate_(gm31_state* state){
  int i; unsigned temp,sum=0,bit=1;
  for(i=0;i<32;i++){
    temp=gm31_CNext(state->xN[i],state->xP[i]);
    state->xP[i]=state->xN[i]; state->xN[i]=temp;
    sum += ((temp<gm31_halfg)?0:bit); bit*=2;
  }
  return sum;
}

extern "C" __device__ __host__ float gm31_generate_uniform_float_(gm31_state* state){
  int i; unsigned temp,sum=0,bit=1;
  for(i=0;i<32;i++){
    temp=gm31_CNext(state->xN[i],state->xP[i]);
    state->xP[i]=state->xN[i]; state->xN[i]=temp;
    sum += ((temp<gm31_halfg)?0:bit); bit*=2;
  }
  return ((float) sum) * 2.3283064365386963e-10;
}

extern "C" __host__ void gm31_print_state_(gm31_state* state){int i;
    printf("Generator State:\nxN={");
    for(i=0;i<32;i++) {printf("%u",state->xN[i]%gm31_g); printf((i<31)?",":"}\nxP={");}
    for(i=0;i<32;i++) {printf("%u",state->xP[i]%gm31_g); printf((i<31)?",":"}\n\n");}
}

extern "C" __host__ void gm31_print_sse_state_(gm31_sse_state* state){int i;
    printf("Generator State:\nxN={");
    for(i=0;i<32;i++) {printf("%u",state->xN[2*i]%gm31_g); printf((i<31)?",":"}\nxP={");}
    for(i=0;i<32;i++) {printf("%u",state->xP[2*i]%gm31_g); printf((i<31)?",":"}\n\n");}
}

__global__ void gm31_kernel_generate_array(gm31_state* state, unsigned int* out, long* length) {
    unsigned temp,i,orbit,seqNum; long offset;

    __shared__ unsigned xP[gm31_THREADS];  // one generator per s=32 threads, i.e. one orbit
    __shared__ unsigned xN[gm31_THREADS];  // per thread, i.e. blockDim.x orbits per block
    __shared__ unsigned  a[gm31_THREADS];  // array "a" contains corresponding parts of output

    orbit   = threadIdx.x % 32;
    seqNum  = (threadIdx.x + blockIdx.x * blockDim.x)>>5;  // RNG_sequence index
    offset  = seqNum*(*length);                            // start of the section in the output array

    xP[threadIdx.x]=gm31_GetNextAny(state->xP[orbit],state->xN[orbit],offset);
    xN[threadIdx.x]=gm31_GetNextAny(state->xP[orbit],state->xN[orbit],offset+1);

    for(i=0;i<(*length);i++){       // each s=32 threads result in "length" values in the output array

      temp = gm31_CNext( xN[threadIdx.x], xP[threadIdx.x] );
      xP[threadIdx.x] = xN[threadIdx.x]; xN[threadIdx.x] = temp; 
      a[threadIdx.x]  = (temp < gm31_halfg ? 0 : (1<<orbit) );

      __syncthreads();
      if((orbit&3)==0) a[threadIdx.x] = a[threadIdx.x]+a[threadIdx.x+1]+a[threadIdx.x+2]+a[threadIdx.x+3];
      __syncthreads();
      if((orbit&15)==0) a[threadIdx.x] = a[threadIdx.x]+a[threadIdx.x+4]+a[threadIdx.x+8]+a[threadIdx.x+12];
      __syncthreads();
      if(orbit==0){ out[offset+i]=a[threadIdx.x]+a[threadIdx.x+16]; }

    }
}

extern "C" __host__ void gm31_generate_gpu_array_(gm31_state* state, unsigned int* dev_out, unsigned int* length){

   long          mylength = (*length)/gm31_ARRAY_SECTIONS;
   gm31_state*   dev_state;
   long*         dev_length;

   if((mylength*gm31_ARRAY_SECTIONS)<(*length)) mylength++;

   gm31_CUDA_CALL(hipMalloc((void**)&dev_state,sizeof(gm31_state)));
   gm31_CUDA_CALL(hipMalloc((void**)&dev_length,sizeof(long)));
   gm31_CUDA_CALL(hipMemcpy(dev_state,state,sizeof(gm31_state),hipMemcpyHostToDevice));
   gm31_CUDA_CALL(hipMemcpy(dev_length,&mylength,sizeof(long),hipMemcpyHostToDevice));

   gm31_kernel_generate_array<<<gm31_BLOCKS,gm31_THREADS>>>(dev_state,dev_out,dev_length);
   gm31_CUDA_CALL(hipGetLastError());
   
   gm31_CUDA_CALL(hipFree(dev_state)); gm31_CUDA_CALL(hipFree(dev_length));

}

__global__ void gm31_kernel_generate_array_float(gm31_state* state, float* out, long* length) {
    unsigned temp,sum,i,orbit,seqNum; long offset;

    __shared__ unsigned xP[gm31_THREADS];  // one generator per s=32 threads, i.e. one orbit
    __shared__ unsigned xN[gm31_THREADS];  // per thread, i.e. blockDim.x orbits per block
    __shared__ unsigned  a[gm31_THREADS];  // array "a" contains corresponding parts of output

    orbit   = threadIdx.x % 32;
    seqNum  = (threadIdx.x + blockIdx.x * blockDim.x)>>5;  // RNG_sequence index
    offset  = seqNum*(*length);                            // start of the section in the output array

    xP[threadIdx.x]=gm31_GetNextAny(state->xP[orbit],state->xN[orbit],offset);
    xN[threadIdx.x]=gm31_GetNextAny(state->xP[orbit],state->xN[orbit],offset+1);

    for(i=0;i<(*length);i++){       // each s=32 threads result in "length" values in the output array

      temp = gm31_CNext( xN[threadIdx.x], xP[threadIdx.x] );
      xP[threadIdx.x] = xN[threadIdx.x]; xN[threadIdx.x] = temp; 
      a[threadIdx.x]  = (temp < gm31_halfg ? 0 : (1<<orbit) );

      __syncthreads();
      if((orbit&3)==0) a[threadIdx.x] = a[threadIdx.x]+a[threadIdx.x+1]+a[threadIdx.x+2]+a[threadIdx.x+3];
      __syncthreads();
      if((orbit&15)==0) a[threadIdx.x] = a[threadIdx.x]+a[threadIdx.x+4]+a[threadIdx.x+8]+a[threadIdx.x+12];
      __syncthreads();
      if(orbit==0){ sum=a[threadIdx.x]+a[threadIdx.x+16];  out[offset+i]=((float)sum) * 2.3283064365386963e-10; }

    }
}

extern "C" __host__ void gm31_generate_gpu_array_float_(gm31_state* state, float* dev_out, unsigned int* length){

   long          mylength = (*length)/gm31_ARRAY_SECTIONS;
   gm31_state*   dev_state;
   long*         dev_length;

   if((mylength*gm31_ARRAY_SECTIONS)<(*length)) mylength++;

   gm31_CUDA_CALL(hipMalloc((void**)&dev_state,sizeof(gm31_state)));
   gm31_CUDA_CALL(hipMalloc((void**)&dev_length,sizeof(long)));
   gm31_CUDA_CALL(hipMemcpy(dev_state,state,sizeof(gm31_state),hipMemcpyHostToDevice));
   gm31_CUDA_CALL(hipMemcpy(dev_length,&mylength,sizeof(long),hipMemcpyHostToDevice));

   gm31_kernel_generate_array_float<<<gm31_BLOCKS,gm31_THREADS>>>(dev_state,dev_out,dev_length);
   gm31_CUDA_CALL(hipGetLastError());
   
   gm31_CUDA_CALL(hipFree(dev_state)); gm31_CUDA_CALL(hipFree(dev_length));

}

__global__ void gm31_kernel_generate_array_double(gm31_state* state, double* out, long* length) {
    unsigned temp,sum,i,orbit,seqNum; long offset;

    __shared__ unsigned xP[gm31_THREADS];  // one generator per s=32 threads, i.e. one orbit
    __shared__ unsigned xN[gm31_THREADS];  // per thread, i.e. blockDim.x orbits per block
    __shared__ unsigned  a[gm31_THREADS];  // array "a" contains corresponding parts of output

    orbit   = threadIdx.x % 32;
    seqNum  = (threadIdx.x + blockIdx.x * blockDim.x)>>5;  // RNG_sequence index
    offset  = seqNum*(*length);                            // start of the section in the output array

    xP[threadIdx.x]=gm31_GetNextAny(state->xP[orbit],state->xN[orbit],offset);
    xN[threadIdx.x]=gm31_GetNextAny(state->xP[orbit],state->xN[orbit],offset+1);

    for(i=0;i<(*length);i++){       // each s=32 threads result in "length" values in the output array

      temp = gm31_CNext( xN[threadIdx.x], xP[threadIdx.x] );
      xP[threadIdx.x] = xN[threadIdx.x]; xN[threadIdx.x] = temp; 
      a[threadIdx.x]  = (temp < gm31_halfg ? 0 : (1<<orbit) );

      __syncthreads();
      if((orbit&3)==0) a[threadIdx.x] = a[threadIdx.x]+a[threadIdx.x+1]+a[threadIdx.x+2]+a[threadIdx.x+3];
      __syncthreads();
      if((orbit&15)==0) a[threadIdx.x] = a[threadIdx.x]+a[threadIdx.x+4]+a[threadIdx.x+8]+a[threadIdx.x+12];
      __syncthreads();
      if(orbit==0){ sum=a[threadIdx.x]+a[threadIdx.x+16];  out[offset+i]=((double)sum) * 2.3283064365386963e-10; }

    }
}

extern "C" __host__ void gm31_generate_gpu_array_double_(gm31_state* state, double* dev_out, unsigned int* length){

   long          mylength = (*length)/gm31_ARRAY_SECTIONS;
   gm31_state*   dev_state;
   long*         dev_length;

   if((mylength*gm31_ARRAY_SECTIONS)<(*length)) mylength++;

   gm31_CUDA_CALL(hipMalloc((void**)&dev_state,sizeof(gm31_state)));
   gm31_CUDA_CALL(hipMalloc((void**)&dev_length,sizeof(long)));
   gm31_CUDA_CALL(hipMemcpy(dev_state,state,sizeof(gm31_state),hipMemcpyHostToDevice));
   gm31_CUDA_CALL(hipMemcpy(dev_length,&mylength,sizeof(long),hipMemcpyHostToDevice));

   gm31_kernel_generate_array_double<<<gm31_BLOCKS,gm31_THREADS>>>(dev_state,dev_out,dev_length);
   gm31_CUDA_CALL(hipGetLastError());
   
   gm31_CUDA_CALL(hipFree(dev_state)); gm31_CUDA_CALL(hipFree(dev_length));

}

extern "C" __host__ void gm31_generate_array_(gm31_state* state, unsigned int* out, unsigned int* length){

   long          mylength = (*length)/gm31_ARRAY_SECTIONS;
   gm31_state*   dev_state;
   unsigned int* dev_out;
   long*         dev_length;

   if((mylength*gm31_ARRAY_SECTIONS)<(*length)) mylength++;

   gm31_CUDA_CALL(hipMalloc((void**)&dev_state,sizeof(gm31_state)));
   gm31_CUDA_CALL(hipMalloc((void**)&dev_out,mylength*gm31_ARRAY_SECTIONS*sizeof(unsigned int)));
   gm31_CUDA_CALL(hipMalloc((void**)&dev_length,sizeof(long)));
   gm31_CUDA_CALL(hipMemcpy(dev_state,state,sizeof(gm31_state),hipMemcpyHostToDevice));
   gm31_CUDA_CALL(hipMemcpy(dev_length,&mylength,sizeof(long),hipMemcpyHostToDevice));

   gm31_kernel_generate_array<<<gm31_BLOCKS,gm31_THREADS>>>(dev_state,dev_out,dev_length);
   gm31_CUDA_CALL(hipGetLastError());
   
   gm31_CUDA_CALL(hipMemcpy(out,dev_out,(*length)*sizeof(unsigned int),hipMemcpyDeviceToHost));
   gm31_CUDA_CALL(hipFree(dev_state)); gm31_CUDA_CALL(hipFree(dev_out)); 
   gm31_CUDA_CALL(hipFree(dev_length));

}
