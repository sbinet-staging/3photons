// (c) Copyright 2013 Lev Barash, Landau Institute for Theoretical Physics, Russian Academy of Sciences
// This is supplement to the paper:
// L.Yu. Barash, L.N. Shchur, "PRAND: GPU accelerated parallel random number generation library: Using most reliable algorithms and applying parallelism of modern GPUs and CPUs".
// e-mail: barash @ itp.ac.ru (remove space)


#include <hip/hip_runtime.h>
#include<stdio.h>

#define gq58x4_CUDA_CALL(x) do { if((x) != hipSuccess) { printf("Error: %s at %s:%d\n",hipGetErrorString(hipGetLastError()),__FILE__,__LINE__); exit(1);}} while(0)

#define gq58x4_BLOCKS  128
#define gq58x4_THREADS 128
#define gq58x4_ARRAY_SECTIONS (gq58x4_BLOCKS*gq58x4_THREADS/8)

#define gq58x4_k 8
#define gq58x4_q 48
#define gq58x4_g       288230374541099008ULL
#define gq58x4_gdiv16  18014398408818688ULL

typedef unsigned long long lt;

typedef struct{
  lt xN[8] __attribute__ ((aligned(16))),
     xP[8] __attribute__ ((aligned(16)));
} gq58x4_state;

typedef gq58x4_state gq58x4_sse_state;

lt gq58x4_sse_Consts[8] __attribute__ ((aligned(16))) =
   {13835057977972752384ULL,13835057977972752384ULL,1610612736ULL,1610612736ULL,
    288230371923853311ULL,288230371923853311ULL,288230374541099008ULL,288230374541099008ULL};

__host__ unsigned int gq58x4_sse_generate_(gq58x4_sse_state* state){
  unsigned output;
  asm volatile("movaps (%3),%%xmm0\n" \

      "movaps (%2),%%xmm1\n" \
      "movaps (%1),%%xmm4\n" \
      "movaps %%xmm4,(%2)\n" \
      "psllq  $3,%%xmm4\n" \
      "paddq  %%xmm0,%%xmm4\n" \
      "psllq  $4,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm4\n" \
      "psllq  $1,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm4\n" \
      "movaps %%xmm4,%%xmm1\n" \
      "psrlq  $58,%%xmm1\n" \
      "psllq  $29,%%xmm1\n" \
      "movaps %%xmm1,%%xmm3\n" \
      "psllq  $1,%%xmm3\n" \
      "paddq  %%xmm1,%%xmm3\n" \
      "psllq  $29,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm4\n" \
      "paddq  %%xmm3,%%xmm4\n" \
      "movaps %%xmm4,%%xmm1\n" \
      "paddq  16(%3),%%xmm1\n" \
      "pshufd $245,%%xmm1,%%xmm3\n" \
      "pcmpgtd 32(%3),%%xmm3\n" \
      "pand    48(%3),%%xmm3\n" \
      "psubq   %%xmm3,%%xmm4\n" \
      "movaps %%xmm4,(%1)\n" \
      "movaps %%xmm4,%%xmm1\n" \
      "paddq  %%xmm4,%%xmm1\n" \
      "paddq  %%xmm4,%%xmm1\n" \
      "psrlq  $29,%%xmm1\n" \
      "paddq  %%xmm1,%%xmm4\n" \


      "movaps 16(%2),%%xmm1\n" \
      "movaps 16(%1),%%xmm5\n" \
      "movaps %%xmm5,16(%2)\n" \
      "psllq  $3,%%xmm5\n" \
      "paddq  %%xmm0,%%xmm5\n" \
      "psllq  $4,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm5\n" \
      "psllq  $1,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm5\n" \
      "movaps %%xmm5,%%xmm1\n" \
      "psrlq  $58,%%xmm1\n" \
      "psllq  $29,%%xmm1\n" \
      "movaps %%xmm1,%%xmm3\n" \
      "psllq  $1,%%xmm3\n" \
      "paddq  %%xmm1,%%xmm3\n" \
      "psllq  $29,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm5\n" \
      "paddq  %%xmm3,%%xmm5\n" \
      "movaps %%xmm5,%%xmm1\n" \
      "paddq  16(%3),%%xmm1\n" \
      "pshufd $245,%%xmm1,%%xmm3\n" \
      "pcmpgtd 32(%3),%%xmm3\n" \
      "pand    48(%3),%%xmm3\n" \
      "psubq   %%xmm3,%%xmm5\n" \
      "movaps %%xmm5,16(%1)\n" \
      "movaps %%xmm5,%%xmm1\n" \
      "paddq  %%xmm5,%%xmm1\n" \
      "paddq  %%xmm5,%%xmm1\n" \
      "psrlq  $29,%%xmm1\n" \
      "paddq  %%xmm1,%%xmm5\n" \

      "movaps 32(%2),%%xmm1\n" \
      "movaps 32(%1),%%xmm6\n" \
      "movaps %%xmm6,32(%2)\n" \
      "psllq  $3,%%xmm6\n" \
      "paddq  %%xmm0,%%xmm6\n" \
      "psllq  $4,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm6\n" \
      "psllq  $1,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm6\n" \
      "movaps %%xmm6,%%xmm1\n" \
      "psrlq  $58,%%xmm1\n" \
      "psllq  $29,%%xmm1\n" \
      "movaps %%xmm1,%%xmm3\n" \
      "psllq  $1,%%xmm3\n" \
      "paddq  %%xmm1,%%xmm3\n" \
      "psllq  $29,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm6\n" \
      "paddq  %%xmm3,%%xmm6\n" \
      "movaps %%xmm6,%%xmm1\n" \
      "paddq  16(%3),%%xmm1\n" \
      "pshufd $245,%%xmm1,%%xmm3\n" \
      "pcmpgtd 32(%3),%%xmm3\n" \
      "pand    48(%3),%%xmm3\n" \
      "psubq   %%xmm3,%%xmm6\n" \
      "movaps %%xmm6,32(%1)\n" \
      "movaps %%xmm6,%%xmm1\n" \
      "paddq  %%xmm6,%%xmm1\n" \
      "paddq  %%xmm6,%%xmm1\n" \
      "psrlq  $29,%%xmm1\n" \
      "paddq  %%xmm1,%%xmm6\n" \

      "movaps 48(%2),%%xmm1\n" \
      "movaps 48(%1),%%xmm7\n" \
      "movaps %%xmm7,48(%2)\n" \
      "psllq  $3,%%xmm7\n" \
      "paddq  %%xmm0,%%xmm7\n" \
      "psllq  $4,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm7\n" \
      "psllq  $1,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm7\n" \
      "movaps %%xmm7,%%xmm1\n" \
      "psrlq  $58,%%xmm1\n" \
      "psllq  $29,%%xmm1\n" \
      "movaps %%xmm1,%%xmm3\n" \
      "psllq  $1,%%xmm3\n" \
      "paddq  %%xmm1,%%xmm3\n" \
      "psllq  $29,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm7\n" \
      "paddq  %%xmm3,%%xmm7\n" \
      "movaps %%xmm7,%%xmm1\n" \
      "paddq  16(%3),%%xmm1\n" \
      "pshufd $245,%%xmm1,%%xmm3\n" \
      "pcmpgtd 32(%3),%%xmm3\n" \
      "pand    48(%3),%%xmm3\n" \
      "psubq   %%xmm3,%%xmm7\n" \
      "movaps %%xmm7,48(%1)\n" \
      "movaps %%xmm7,%%xmm1\n" \
      "paddq  %%xmm7,%%xmm1\n" \
      "paddq  %%xmm7,%%xmm1\n" \
      "psrlq  $29,%%xmm1\n" \
      "paddq  %%xmm1,%%xmm7\n" \

      "psrlq  $54,%%xmm4\n" \
      "psrlq  $54,%%xmm5\n" \
      "psrlq  $54,%%xmm6\n" \
      "psrlq  $54,%%xmm7\n" \
      "packssdw  %%xmm5,%%xmm4\n" \
      "packssdw  %%xmm7,%%xmm6\n" \
      "packssdw  %%xmm6,%%xmm4\n" \
      "packsswb  %%xmm4,%%xmm4\n" \
      "movaps  %%xmm4,%%xmm0\n" \
      "psrldq   $4,%%xmm0\n" \
      "pslld    $4,%%xmm0\n" \
      "pxor    %%xmm0,%%xmm4\n"
      "movd    %%xmm4,%0\n" \
      "":"=&r"(output):"r"(state->xN),"r"(state->xP),"r"(gq58x4_sse_Consts));
      return output;
}

__device__ __host__ void gq58x4_get_sse_state_(gq58x4_state* state,gq58x4_sse_state* sse_state){
  int i; for(i=0;i<8;i++) {sse_state->xN[i]=state->xN[i]; sse_state->xP[i]=state->xP[i];}
}


__device__ __host__ lt gq58x4_mod_g(lt x){ // returns x (mod g)
  lt F,G; F = (x>>58); G = x-(F<<58)+(F<<29)+(F<<30);
  return ((G>=gq58x4_g) ? (G-gq58x4_g) : G);
}

__device__ __host__ lt gq58x4_MyMult(lt A,lt B){ // returns AB (mod gq58x4_g), where it is implied that A,B<gq58x4_g;
  lt A1,A0,B1,B0,curr,x,m;
  A1=A>>32; B1=B>>32; A0=A-(A1<<32)+(12*A1); B0=B-(B1<<32)+(12*B1);
  if(A0>>32) {A0-=4294967284ULL; A1++;}
  if(B0>>32) {B0-=4294967284ULL; B1++;}
  curr=A1*B0+B1*A0; m=curr>>26; x=curr-(m<<26);
  curr=((3*m+(x<<4))<<28)+(gq58x4_g-12*x)+(144*A1*B1)+(gq58x4_mod_g(A0*B0));
  return gq58x4_mod_g(curr);
}

__device__ __host__ lt gq58x4_CNext2(lt N,lt P,lt myk,lt myq){   // returns (myk*N-myq*P) (mod gq58x4_g)
  lt curr1,curr2;
  curr1=gq58x4_MyMult(myk,N); curr2=gq58x4_MyMult(myq,P);
  if(curr1>=curr2) return (curr1-curr2); else return (gq58x4_g+curr1-curr2);
}

__device__ __host__ lt gq58x4_CNext(lt N,lt P){ // returns (8N-48P) (mod gq58x4_g)
  return gq58x4_mod_g((N+6*(gq58x4_g-P))<<3);
}


__device__ __host__ lt gq58x4_GetNextN(lt x0,lt x1,unsigned int n){ //returns x_{2^n}
  lt myk=gq58x4_k,myq=gq58x4_q,i,x=x1;
  for(i=0;i<n;i++){
    x=gq58x4_CNext2(x,x0,myk,myq);
    myk=gq58x4_CNext2(myk,2,myk,myq);
    myq=gq58x4_CNext2(myq,0,myq,0);
  }
  return x;
}

__device__ __host__ lt gq58x4_GetNextAny(lt x0,lt x1,lt N64,lt N0){ //N=2^64*N64+N0+1
  lt i,xp=x0,xn=x1,xpnew,xnnew,shift=0;
  i=N0; while(i>0){
    if(i%2==1){                        // xp,xn ----> 2^shift
      xpnew=gq58x4_GetNextN(xp,xn,shift);
      xnnew=gq58x4_GetNextN(xn,gq58x4_CNext(xn,xp),shift);
      xp=xpnew; xn=xnnew;
    }
    i/=2; shift++;
  }
  i=N64; shift=64; while(i>0){
    if(i%2==1){                        // xp,xn ----> 2^shift
      xpnew=gq58x4_GetNextN(xp,xn,shift);
      xnnew=gq58x4_GetNextN(xn,gq58x4_CNext(xn,xp),shift);
      xp=xpnew; xn=xnnew;
    }
    i/=2; shift++;
  }
  return xp;                       // returns x_N, where N=2^64*N64+N0+1
}

__device__ __host__ void gq58x4_skipahead_(gq58x4_state* state, lt offset64, lt offset0){ // offset=offset64*2^64+offset0+1
  lt xn,xp,j; 
  for(j=0;j<8;j++){
    xp=gq58x4_GetNextAny(state->xP[j],state->xN[j],offset64,offset0);
    xn=gq58x4_GetNextAny(state->xP[j],state->xN[j],offset64,offset0+1);
    state->xP[j]=xp; state->xN[j]=xn;
  }
}

__device__ __host__ void gq58x4_init_(gq58x4_state* state){
  lt x0=100152853817629549ULL,x1=132388305121829306ULL,xp,xn,j;
  for(j=0;j<8;j++){
    xp=gq58x4_GetNextAny(x0,x1,0,35048736516210783ULL);
    xn=gq58x4_GetNextAny(x0,x1,0,35048736516210784ULL);
    state->xP[j]=xp; state->xN[j]=xn; x0=xp; x1=xn;
  }
}

__device__ __host__ void gq58x4_init_short_sequence_(gq58x4_state* state,unsigned SequenceNumber){
  gq58x4_init_(state);                     // 0 <= SequenceNumber < 3*10^8;   length of each sequence <= 8*10^7
  gq58x4_skipahead_(state,0,82927047ULL*(unsigned long long)SequenceNumber);
}

__device__ __host__ void gq58x4_init_medium_sequence_(gq58x4_state* state,unsigned SequenceNumber){
  gq58x4_init_(state);                     // 0 <= SequenceNumber < 3*10^6;   length of each sequence <= 8*10^9
  gq58x4_skipahead_(state,0,8799201913ULL*(unsigned long long)SequenceNumber);
}

__device__ __host__ void gq58x4_init_long_sequence_(gq58x4_state* state,unsigned SequenceNumber){
  gq58x4_init_(state);                     // 0 <= SequenceNumber < 3*10^4;   length of each sequence <= 8*10^11
  gq58x4_skipahead_(state,0,828317697521ULL*(unsigned long long)SequenceNumber);
}

__device__ __host__ unsigned int gq58x4_generate_(gq58x4_state* state){
  unsigned int sum=0; int i; lt temp;
  for(i=0;i<8;i++){ 
    temp=gq58x4_mod_g((state->xN[i]+6*(gq58x4_g-state->xP[i]))<<3);
    state->xP[i]=state->xN[i]; state->xN[i]=temp;
    sum+= ((temp/gq58x4_gdiv16)<<((i<4)?(8*i):(8*i-28)));
  }
  return sum;
}

__device__ __host__ float gq58x4_generate_uniform_float_(gq58x4_state* state){
  unsigned int sum=0; int i; lt temp;
  for(i=0;i<8;i++){ 
    temp=gq58x4_mod_g((state->xN[i]+6*(gq58x4_g-state->xP[i]))<<3);
    state->xP[i]=state->xN[i]; state->xN[i]=temp;
    sum+= ((temp/gq58x4_gdiv16)<<((i<4)?(8*i):(8*i-28)));
  }
  return ((float) sum) * 2.3283064365386963e-10;
}

__host__ void gq58x4_print_state_(gq58x4_state* state){int i;
    printf("Generator State:\nxN={");
    for(i=0;i<8;i++) {printf("%llu",state->xN[i]%gq58x4_g); printf((i<7)?",":"}\nxP={");}
    for(i=0;i<8;i++) {printf("%llu",state->xP[i]%gq58x4_g); printf((i<7)?",":"}\n\n");}
}

__host__ void gq58x4_print_sse_state_(gq58x4_sse_state* state){int i;
    printf("Generator State:\nxN={");
    for(i=0;i<8;i++) {printf("%llu",state->xN[i]%gq58x4_g); printf((i<7)?",":"}\nxP={");}
    for(i=0;i<8;i++) {printf("%llu",state->xP[i]%gq58x4_g); printf((i<7)?",":"}\n\n");}
}

__global__ void gq58x4_kernel_generate_array(gq58x4_state* state, unsigned int* out, long* length) {
    unsigned sum,i,j,orbit,seqNum,shift; long offset; lt temp;

    __shared__ lt xP[gq58x4_THREADS];        // one generator per s=8 threads, i.e. one orbit
    __shared__ lt xN[gq58x4_THREADS];        // per thread, i.e. blockDim.x orbits per block
    __shared__ unsigned  a[gq58x4_THREADS];  // array "a" contains corresponding parts of output

    orbit   = threadIdx.x % 8;
    seqNum  = (threadIdx.x + blockIdx.x * blockDim.x)>>3;  // RNG_sequence index
    offset  = seqNum*(*length);                            // start of the section in the output array

    xP[threadIdx.x]=gq58x4_GetNextAny(state->xP[orbit],state->xN[orbit],0,offset);
    xN[threadIdx.x]=gq58x4_GetNextAny(state->xP[orbit],state->xN[orbit],0,offset+1);

    shift=((orbit<4)?(8*orbit):(8*orbit-28));

    for(i=0;i<(*length);i++){

      temp = gq58x4_CNext( xN[threadIdx.x], xP[threadIdx.x] );
      xP[threadIdx.x] = xN[threadIdx.x]; xN[threadIdx.x] = temp; 
      a[threadIdx.x]  = ((temp/gq58x4_gdiv16)<<shift);

      __syncthreads();              // each s=8 threads result in "length" values in the output array

      if(orbit==0){ sum=0; for(j=0;j<8;j++) sum+=a[threadIdx.x+j]; out[offset+i]=sum; }

    }
}

__host__ void gq58x4_generate_gpu_array_(gq58x4_state* state, unsigned int* dev_out, long length){

   long          mylength = length/gq58x4_ARRAY_SECTIONS;
   gq58x4_state*   dev_state;
   long*         dev_length;

   if((mylength*gq58x4_ARRAY_SECTIONS)<length) mylength++;

   gq58x4_CUDA_CALL(hipMalloc((void**)&dev_state,sizeof(gq58x4_state)));
   gq58x4_CUDA_CALL(hipMalloc((void**)&dev_length,sizeof(long)));
   gq58x4_CUDA_CALL(hipMemcpy(dev_state,state,sizeof(gq58x4_state),hipMemcpyHostToDevice));
   gq58x4_CUDA_CALL(hipMemcpy(dev_length,&mylength,sizeof(long),hipMemcpyHostToDevice));

   gq58x4_kernel_generate_array<<<gq58x4_BLOCKS,gq58x4_THREADS>>>(dev_state,dev_out,dev_length);
   gq58x4_CUDA_CALL(hipGetLastError());
   
   gq58x4_CUDA_CALL(hipFree(dev_state)); gq58x4_CUDA_CALL(hipFree(dev_length));

}

__global__ void gq58x4_kernel_generate_array_float(gq58x4_state* state, float* out, long* length) {
    unsigned sum,i,j,orbit,seqNum,shift; long offset; lt temp;

    __shared__ lt xP[gq58x4_THREADS];        // one generator per s=8 threads, i.e. one orbit
    __shared__ lt xN[gq58x4_THREADS];        // per thread, i.e. blockDim.x orbits per block
    __shared__ unsigned  a[gq58x4_THREADS];  // array "a" contains corresponding parts of output

    orbit   = threadIdx.x % 8;
    seqNum  = (threadIdx.x + blockIdx.x * blockDim.x)>>3;  // RNG_sequence index
    offset  = seqNum*(*length);                            // start of the section in the output array

    xP[threadIdx.x]=gq58x4_GetNextAny(state->xP[orbit],state->xN[orbit],0,offset);
    xN[threadIdx.x]=gq58x4_GetNextAny(state->xP[orbit],state->xN[orbit],0,offset+1);

    shift=((orbit<4)?(8*orbit):(8*orbit-28));

    for(i=0;i<(*length);i++){

      temp = gq58x4_CNext( xN[threadIdx.x], xP[threadIdx.x] );
      xP[threadIdx.x] = xN[threadIdx.x]; xN[threadIdx.x] = temp; 
      a[threadIdx.x]  = ((temp/gq58x4_gdiv16)<<shift);

      __syncthreads();              // each s=8 threads result in "length" values in the output array

      if(orbit==0){ sum=0; for(j=0;j<8;j++) sum+=a[threadIdx.x+j]; out[offset+i]=((float)sum)* 2.3283064365386963e-10; }

    }
}

__host__ void gq58x4_generate_gpu_array_float_(gq58x4_state* state, float* dev_out, long length){

   long          mylength = length/gq58x4_ARRAY_SECTIONS;
   gq58x4_state*   dev_state;
   long*         dev_length;

   if((mylength*gq58x4_ARRAY_SECTIONS)<length) mylength++;

   gq58x4_CUDA_CALL(hipMalloc((void**)&dev_state,sizeof(gq58x4_state)));
   gq58x4_CUDA_CALL(hipMalloc((void**)&dev_length,sizeof(long)));
   gq58x4_CUDA_CALL(hipMemcpy(dev_state,state,sizeof(gq58x4_state),hipMemcpyHostToDevice));
   gq58x4_CUDA_CALL(hipMemcpy(dev_length,&mylength,sizeof(long),hipMemcpyHostToDevice));

   gq58x4_kernel_generate_array_float<<<gq58x4_BLOCKS,gq58x4_THREADS>>>(dev_state,dev_out,dev_length);
   gq58x4_CUDA_CALL(hipGetLastError());
   
   gq58x4_CUDA_CALL(hipFree(dev_state)); gq58x4_CUDA_CALL(hipFree(dev_length));

}

__global__ void gq58x4_kernel_generate_array_double(gq58x4_state* state, double* out, long* length) {
    unsigned sum,i,j,orbit,seqNum,shift; long offset; lt temp;

    __shared__ lt xP[gq58x4_THREADS];        // one generator per s=8 threads, i.e. one orbit
    __shared__ lt xN[gq58x4_THREADS];        // per thread, i.e. blockDim.x orbits per block
    __shared__ unsigned  a[gq58x4_THREADS];  // array "a" contains corresponding parts of output

    orbit   = threadIdx.x % 8;
    seqNum  = (threadIdx.x + blockIdx.x * blockDim.x)>>3;  // RNG_sequence index
    offset  = seqNum*(*length);                            // start of the section in the output array

    xP[threadIdx.x]=gq58x4_GetNextAny(state->xP[orbit],state->xN[orbit],0,offset);
    xN[threadIdx.x]=gq58x4_GetNextAny(state->xP[orbit],state->xN[orbit],0,offset+1);

    shift=((orbit<4)?(8*orbit):(8*orbit-28));

    for(i=0;i<(*length);i++){

      temp = gq58x4_CNext( xN[threadIdx.x], xP[threadIdx.x] );
      xP[threadIdx.x] = xN[threadIdx.x]; xN[threadIdx.x] = temp; 
      a[threadIdx.x]  = ((temp/gq58x4_gdiv16)<<shift);

      __syncthreads();              // each s=8 threads result in "length" values in the output array

      if(orbit==0){ sum=0; for(j=0;j<8;j++) sum+=a[threadIdx.x+j]; out[offset+i]=((double)sum)* 2.3283064365386963e-10; }

    }
}

__host__ void gq58x4_generate_gpu_array_double_(gq58x4_state* state, double* dev_out, long length){

   long          mylength = length/gq58x4_ARRAY_SECTIONS;
   gq58x4_state*   dev_state;
   long*         dev_length;

   if((mylength*gq58x4_ARRAY_SECTIONS)<length) mylength++;

   gq58x4_CUDA_CALL(hipMalloc((void**)&dev_state,sizeof(gq58x4_state)));
   gq58x4_CUDA_CALL(hipMalloc((void**)&dev_length,sizeof(long)));
   gq58x4_CUDA_CALL(hipMemcpy(dev_state,state,sizeof(gq58x4_state),hipMemcpyHostToDevice));
   gq58x4_CUDA_CALL(hipMemcpy(dev_length,&mylength,sizeof(long),hipMemcpyHostToDevice));

   gq58x4_kernel_generate_array_double<<<gq58x4_BLOCKS,gq58x4_THREADS>>>(dev_state,dev_out,dev_length);
   gq58x4_CUDA_CALL(hipGetLastError());
   
   gq58x4_CUDA_CALL(hipFree(dev_state)); gq58x4_CUDA_CALL(hipFree(dev_length));

}

__host__ void gq58x4_generate_array_(gq58x4_state* state, unsigned int* out, long length){

   long          mylength = length/gq58x4_ARRAY_SECTIONS;
   gq58x4_state*   dev_state;
   unsigned int* dev_out;
   long*         dev_length;

   if((mylength*gq58x4_ARRAY_SECTIONS)<length) mylength++;

   gq58x4_CUDA_CALL(hipMalloc((void**)&dev_state,sizeof(gq58x4_state)));
   gq58x4_CUDA_CALL(hipMalloc((void**)&dev_out,mylength*gq58x4_ARRAY_SECTIONS*sizeof(unsigned int)));
   gq58x4_CUDA_CALL(hipMalloc((void**)&dev_length,sizeof(long)));
   gq58x4_CUDA_CALL(hipMemcpy(dev_state,state,sizeof(gq58x4_state),hipMemcpyHostToDevice));
   gq58x4_CUDA_CALL(hipMemcpy(dev_length,&mylength,sizeof(long),hipMemcpyHostToDevice));

   gq58x4_kernel_generate_array<<<gq58x4_BLOCKS,gq58x4_THREADS>>>(dev_state,dev_out,dev_length);
   gq58x4_CUDA_CALL(hipGetLastError());
   
   gq58x4_CUDA_CALL(hipMemcpy(out,dev_out,length*sizeof(unsigned int),hipMemcpyDeviceToHost));
   gq58x4_CUDA_CALL(hipFree(dev_state)); gq58x4_CUDA_CALL(hipFree(dev_out)); 
   gq58x4_CUDA_CALL(hipFree(dev_length));

}