// (c) Copyright 2013 Lev Barash, Landau Institute for Theoretical Physics, Russian Academy of Sciences
// This is supplement to the paper:
// L.Yu. Barash, L.N. Shchur, "PRAND: GPU accelerated parallel random number generation library: Using most reliable algorithms and applying parallelism of modern GPUs and CPUs".
// e-mail: barash @ itp.ac.ru (remove space)


#include <hip/hip_runtime.h>
#include<stdio.h>

#define gq58x3_CUDA_CALL(x) do { if((x) != hipSuccess) { printf("Error: %s at %s:%d\n",hipGetErrorString(hipGetLastError()),__FILE__,__LINE__); exit(1);}} while(0)

#define gq58x3_BLOCKS  128
#define gq58x3_THREADS 192
#define gq58x3_ARRAY_SECTIONS (gq58x3_BLOCKS*gq58x3_THREADS/12)

#define gq58x3_k       8
#define gq58x3_q       48
#define gq58x3_g       288230374541099008ULL
#define gq58x3_gdiv8   36028796817637376ULL

typedef unsigned long long lt;

typedef struct{
  lt xN[12] __attribute__ ((aligned(16))),
     xP[12] __attribute__ ((aligned(16)));
} gq58x3_state;

typedef gq58x3_state gq58x3_sse_state;

lt gq58x3_sse_Consts[10] __attribute__ ((aligned(16))) =
   {13835057977972752384ULL,13835057977972752384ULL,1610612736ULL,1610612736ULL,
    288230371923853311ULL,288230371923853311ULL,288230374541099008ULL,288230374541099008ULL,
    18157383382357244923ULL,18157383382357244923ULL}; 

__host__ unsigned int gq58x3_sse_generate_(gq58x3_sse_state* state){
  unsigned output;
    asm volatile("movaps (%3),%%xmm0\n" \

      "movaps (%2),%%xmm1\n" \
      "movaps (%1),%%xmm4\n" \
      "movaps %%xmm4,(%2)\n" \
      "psllq  $3,%%xmm4\n" \
      "paddq  %%xmm0,%%xmm4\n" \
      "psllq  $4,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm4\n" \
      "psllq  $1,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm4\n" \
      "movaps %%xmm4,%%xmm1\n" \
      "psrlq  $58,%%xmm1\n" \
      "psllq  $29,%%xmm1\n" \
      "movaps %%xmm1,%%xmm3\n" \
      "psllq  $1,%%xmm3\n" \
      "paddq  %%xmm1,%%xmm3\n" \
      "psllq  $29,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm4\n" \
      "paddq  %%xmm3,%%xmm4\n" \
      "movaps %%xmm4,%%xmm1\n" \
      "paddq  16(%3),%%xmm1\n" \
      "pshufd $245,%%xmm1,%%xmm3\n" \
      "pcmpgtd 32(%3),%%xmm3\n" \
      "pand    48(%3),%%xmm3\n" \
      "psubq   %%xmm3,%%xmm4\n" \
      "movaps %%xmm4,(%1)\n" \
      "movaps %%xmm4,%%xmm1\n" \
      "paddq  %%xmm4,%%xmm1\n" \
      "paddq  %%xmm4,%%xmm1\n" \
      "psrlq  $29,%%xmm1\n" \
      "paddq  %%xmm1,%%xmm4\n" \

      "movaps 16(%2),%%xmm1\n" \
      "movaps 16(%1),%%xmm5\n" \
      "movaps %%xmm5,16(%2)\n" \
      "psllq  $3,%%xmm5\n" \
      "paddq  %%xmm0,%%xmm5\n" \
      "psllq  $4,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm5\n" \
      "psllq  $1,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm5\n" \
      "movaps %%xmm5,%%xmm1\n" \
      "psrlq  $58,%%xmm1\n" \
      "psllq  $29,%%xmm1\n" \
      "movaps %%xmm1,%%xmm3\n" \
      "psllq  $1,%%xmm3\n" \
      "paddq  %%xmm1,%%xmm3\n" \
      "psllq  $29,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm5\n" \
      "paddq  %%xmm3,%%xmm5\n" \
      "movaps %%xmm5,%%xmm1\n" \
      "paddq  16(%3),%%xmm1\n" \
      "pshufd $245,%%xmm1,%%xmm3\n" \
      "pcmpgtd 32(%3),%%xmm3\n" \
      "pand    48(%3),%%xmm3\n" \
      "psubq   %%xmm3,%%xmm5\n" \
      "movaps %%xmm5,16(%1)\n" \
      "movaps %%xmm5,%%xmm1\n" \
      "paddq  %%xmm5,%%xmm1\n" \
      "paddq  %%xmm5,%%xmm1\n" \
      "psrlq  $29,%%xmm1\n" \
      "paddq  %%xmm1,%%xmm5\n" \

      "movaps 32(%2),%%xmm1\n" \
      "movaps 32(%1),%%xmm6\n" \
      "movaps %%xmm6,32(%2)\n" \
      "psllq  $3,%%xmm6\n" \
      "paddq  %%xmm0,%%xmm6\n" \
      "psllq  $4,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm6\n" \
      "psllq  $1,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm6\n" \
      "movaps %%xmm6,%%xmm1\n" \
      "psrlq  $58,%%xmm1\n" \
      "psllq  $29,%%xmm1\n" \
      "movaps %%xmm1,%%xmm3\n" \
      "psllq  $1,%%xmm3\n" \
      "paddq  %%xmm1,%%xmm3\n" \
      "psllq  $29,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm6\n" \
      "paddq  %%xmm3,%%xmm6\n" \
      "movaps %%xmm6,%%xmm1\n" \
      "paddq  16(%3),%%xmm1\n" \
      "pshufd $245,%%xmm1,%%xmm3\n" \
      "pcmpgtd 32(%3),%%xmm3\n" \
      "pand    48(%3),%%xmm3\n" \
      "psubq   %%xmm3,%%xmm6\n" \
      "movaps %%xmm6,32(%1)\n" \
      "movaps %%xmm6,%%xmm1\n" \
      "paddq  %%xmm6,%%xmm1\n" \
      "paddq  %%xmm6,%%xmm1\n" \
      "psrlq  $29,%%xmm1\n" \
      "paddq  %%xmm1,%%xmm6\n" \

      "movaps 48(%2),%%xmm1\n" \
      "movaps 48(%1),%%xmm7\n" \
      "movaps %%xmm7,48(%2)\n" \
      "psllq  $3,%%xmm7\n" \
      "paddq  %%xmm0,%%xmm7\n" \
      "psllq  $4,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm7\n" \
      "psllq  $1,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm7\n" \
      "movaps %%xmm7,%%xmm1\n" \
      "psrlq  $58,%%xmm1\n" \
      "psllq  $29,%%xmm1\n" \
      "movaps %%xmm1,%%xmm3\n" \
      "psllq  $1,%%xmm3\n" \
      "paddq  %%xmm1,%%xmm3\n" \
      "psllq  $29,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm7\n" \
      "paddq  %%xmm3,%%xmm7\n" \
      "movaps %%xmm7,%%xmm1\n" \
      "paddq  16(%3),%%xmm1\n" \
      "pshufd $245,%%xmm1,%%xmm3\n" \
      "pcmpgtd 32(%3),%%xmm3\n" \
      "pand    48(%3),%%xmm3\n" \
      "psubq   %%xmm3,%%xmm7\n" \
      "movaps %%xmm7,48(%1)\n" \
      "movaps %%xmm7,%%xmm1\n" \
      "paddq  %%xmm7,%%xmm1\n" \
      "paddq  %%xmm7,%%xmm1\n" \
      "psrlq  $29,%%xmm1\n" \
      "paddq  %%xmm1,%%xmm7\n" \

      "psrlq  $55,%%xmm4\n" \
      "psrlq  $55,%%xmm5\n" \
      "psrlq  $55,%%xmm6\n" \
      "psrlq  $55,%%xmm7\n" \
      "packssdw  %%xmm5,%%xmm4\n" \
      "packssdw  %%xmm7,%%xmm6\n" \

      "movaps 64(%2),%%xmm1\n" \
      "movaps 64(%1),%%xmm5\n" \
      "movaps %%xmm5,64(%2)\n" \
      "psllq  $3,%%xmm5\n" \
      "paddq  %%xmm0,%%xmm5\n" \
      "psllq  $4,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm5\n" \
      "psllq  $1,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm5\n" \
      "movaps %%xmm5,%%xmm1\n" \
      "psrlq  $58,%%xmm1\n" \
      "psllq  $29,%%xmm1\n" \
      "movaps %%xmm1,%%xmm3\n" \
      "psllq  $1,%%xmm3\n" \
      "paddq  %%xmm1,%%xmm3\n" \
      "psllq  $29,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm5\n" \
      "paddq  %%xmm3,%%xmm5\n" \
      "movaps %%xmm5,%%xmm1\n" \
      "paddq  16(%3),%%xmm1\n" \
      "pshufd $245,%%xmm1,%%xmm3\n" \
      "pcmpgtd 32(%3),%%xmm3\n" \
      "pand    48(%3),%%xmm3\n" \
      "psubq   %%xmm3,%%xmm5\n" \
      "movaps %%xmm5,64(%1)\n" \
      "movaps %%xmm5,%%xmm1\n" \
      "paddq  %%xmm5,%%xmm1\n" \
      "paddq  %%xmm5,%%xmm1\n" \
      "psrlq  $29,%%xmm1\n" \
      "paddq  %%xmm1,%%xmm5\n" \

      "movaps 80(%2),%%xmm1\n" \
      "movaps 80(%1),%%xmm7\n" \
      "movaps %%xmm7,80(%2)\n" \
      "psllq  $3,%%xmm7\n" \
      "paddq  %%xmm0,%%xmm7\n" \
      "psllq  $4,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm7\n" \
      "psllq  $1,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm7\n" \
      "movaps %%xmm7,%%xmm1\n" \
      "psrlq  $58,%%xmm1\n" \
      "psllq  $29,%%xmm1\n" \
      "movaps %%xmm1,%%xmm3\n" \
      "psllq  $1,%%xmm3\n" \
      "paddq  %%xmm1,%%xmm3\n" \
      "psllq  $29,%%xmm1\n" \
      "psubq  %%xmm1,%%xmm7\n" \
      "paddq  %%xmm3,%%xmm7\n" \
      "movaps %%xmm7,%%xmm1\n" \
      "paddq  16(%3),%%xmm1\n" \
      "pshufd $245,%%xmm1,%%xmm3\n" \
      "pcmpgtd 32(%3),%%xmm3\n" \
      "pand    48(%3),%%xmm3\n" \
      "psubq   %%xmm3,%%xmm7\n" \
      "movaps %%xmm7,80(%1)\n" \
      "movaps %%xmm7,%%xmm1\n" \
      "paddq  %%xmm7,%%xmm1\n" \
      "paddq  %%xmm7,%%xmm1\n" \
      "psrlq  $29,%%xmm1\n" \
      "paddq  %%xmm1,%%xmm7\n" \

      "psrlq  $55,%%xmm5\n" \
      "psrlq  $55,%%xmm7\n" \
      "packssdw  %%xmm7,%%xmm5\n" \

      "packssdw  %%xmm4,%%xmm4\n" \
      "packssdw  %%xmm6,%%xmm6\n" \
      "packssdw  %%xmm5,%%xmm5\n" \
      "packsswb  %%xmm4,%%xmm4\n" \
      "packsswb  %%xmm6,%%xmm6\n" \
      "packsswb  %%xmm5,%%xmm5\n" \
      "pand   64(%3),%%xmm4\n" \
      "pslld  $6,%%xmm4\n" \
      "pxor   %%xmm4,%%xmm5\n" \
      "pslld  $3,%%xmm6\n" \
      "pxor   %%xmm6,%%xmm5\n" \
      "movd    %%xmm5,%0\n" \

      "":"=&r"(output):"r"(state->xN),"r"(state->xP),"r"(gq58x3_sse_Consts));
    return output;
}

__device__ __host__ void gq58x3_get_sse_state_(gq58x3_state* state,gq58x3_sse_state* sse_state){
  int i; for(i=0;i<12;i++) {sse_state->xN[i]=state->xN[i]; sse_state->xP[i]=state->xP[i];}
}

__device__ __host__ lt gq58x3_mod_g(lt x){ // returns x (mod g)
  lt F,G; F = (x>>58); G = x-(F<<58)+(F<<29)+(F<<30);
  return ((G>=gq58x3_g) ? (G-gq58x3_g) : G);
}

__device__ __host__ lt gq58x3_MyMult(lt A,lt B){ // returns AB (mod gq58x3_g), where it is implied that A,B<gq58x3_g;
  lt A1,A0,B1,B0,curr,x,m;
  A1=A>>32; B1=B>>32; A0=A-(A1<<32)+(12*A1); B0=B-(B1<<32)+(12*B1);
  if(A0>>32) {A0-=4294967284ULL; A1++;}
  if(B0>>32) {B0-=4294967284ULL; B1++;}
  curr=A1*B0+B1*A0; m=curr>>26; x=curr-(m<<26);
  curr=((3*m+(x<<4))<<28)+(gq58x3_g-12*x)+(144*A1*B1)+(gq58x3_mod_g(A0*B0));
  return gq58x3_mod_g(curr);
}

__device__ __host__ lt gq58x3_CNext2(lt N,lt P,lt myk,lt myq){   // returns (myk*N-myq*P) (mod gq58x3_g)
  lt curr1,curr2;
  curr1=gq58x3_MyMult(myk,N); curr2=gq58x3_MyMult(myq,P);
  if(curr1>=curr2) return (curr1-curr2); else return (gq58x3_g+curr1-curr2);
}

__device__ __host__ lt gq58x3_CNext(lt N,lt P){ // returns (8N-48P) (mod gq58x3_g)
  return gq58x3_mod_g((N+6*(gq58x3_g-P))<<3);
}

__device__ __host__ lt gq58x3_GetNextN(lt x0,lt x1,unsigned int n){ //returns x_{2^n}
  lt myk=gq58x3_k,myq=gq58x3_q,i,x=x1;
  for(i=0;i<n;i++){
    x=gq58x3_CNext2(x,x0,myk,myq);
    myk=gq58x3_CNext2(myk,2,myk,myq);
    myq=gq58x3_CNext2(myq,0,myq,0);
  }
  return x;
}

__device__ __host__ lt gq58x3_GetNextAny(lt x0,lt x1,lt N64,lt N0){ //N=2^64*N64+N0+1
  lt i,xp=x0,xn=x1,xpnew,xnnew,shift=0;
  i=N0; while(i>0){
    if(i%2==1){                        // xp,xn ----> 2^shift
      xpnew=gq58x3_GetNextN(xp,xn,shift);
      xnnew=gq58x3_GetNextN(xn,gq58x3_CNext(xn,xp),shift);
      xp=xpnew; xn=xnnew;
    }
    i/=2; shift++;
  }
  i=N64; shift=64; while(i>0){
    if(i%2==1){                        // xp,xn ----> 2^shift
      xpnew=gq58x3_GetNextN(xp,xn,shift);
      xnnew=gq58x3_GetNextN(xn,gq58x3_CNext(xn,xp),shift);
      xp=xpnew; xn=xnnew;
    }
    i/=2; shift++;
  }
  return xp;                       // returns x_N, where N=2^64*N64+N0+1
}

__device__ __host__ void gq58x3_skipahead_(gq58x3_state* state, lt offset64, lt offset0){ // offset=offset64*2^64+offset0+1
  lt xn,xp,j; 
  for(j=0;j<12;j++){
    xp=gq58x3_GetNextAny(state->xP[j],state->xN[j],offset64,offset0);
    xn=gq58x3_GetNextAny(state->xP[j],state->xN[j],offset64,offset0+1);
    state->xP[j]=xp; state->xN[j]=xn;
  }
}

__device__ __host__ void gq58x3_init_(gq58x3_state* state){
  lt x0=100142853817629549ULL,x1=133388305121829306ULL,xp,xn,j;
  for(j=0;j<12;j++){
    xp=gq58x3_GetNextAny(x0,x1,0,24014539279611495ULL);
    xn=gq58x3_GetNextAny(x0,x1,0,24014539279611496ULL);
    state->xP[j]=xp; state->xN[j]=xn; x0=xp; x1=xn;
  }
}

__device__ __host__ void gq58x3_init_short_sequence_(gq58x3_state* state,unsigned SequenceNumber){
  gq58x3_init_(state);                     // 0 <= SequenceNumber < 2*10^8;   length of each sequence <= 8*10^7
  gq58x3_skipahead_(state,0,82927047ULL*(unsigned long long)SequenceNumber);
}

__device__ __host__ void gq58x3_init_medium_sequence_(gq58x3_state* state,unsigned SequenceNumber){
  gq58x3_init_(state);                     // 0 <= SequenceNumber < 2*10^6;   length of each sequence <= 8*10^9
  gq58x3_skipahead_(state,0,8799201913ULL*(unsigned long long)SequenceNumber);
}

__device__ __host__ void gq58x3_init_long_sequence_(gq58x3_state* state,unsigned SequenceNumber){
  gq58x3_init_(state);                     // 0 <= SequenceNumber < 2*10^4;   length of each sequence <= 8*10^11
  gq58x3_skipahead_(state,0,828317697521ULL*(unsigned long long)SequenceNumber);
}

__device__ __host__ unsigned int gq58x3_generate_(gq58x3_state* state){
  unsigned sum=0; int i; lt temp;
  for(i=0;i<12;i++){
    temp=gq58x3_mod_g((state->xN[i]+6*(gq58x3_g-state->xP[i]))<<3);
    state->xP[i]=state->xN[i]; state->xN[i]=temp; 
    sum+=((((temp/gq58x3_gdiv8)<<((i<4)?6:((i<8)?3:0)))%256)<<(8*(i%4)));
  }
  return sum;
}

__device__ __host__ float gq58x3_generate_uniform_float_(gq58x3_state* state){
  unsigned sum=0; int i; lt temp;
  for(i=0;i<12;i++){
    temp=gq58x3_mod_g((state->xN[i]+6*(gq58x3_g-state->xP[i]))<<3);
    state->xP[i]=state->xN[i]; state->xN[i]=temp; 
    sum+=((((temp/gq58x3_gdiv8)<<((i<4)?6:((i<8)?3:0)))%256)<<(8*(i%4)));
  }
  return ((float) sum) * 2.3283064365386963e-10;
}

__host__ void gq58x3_print_state_(gq58x3_state* state){int i;
    printf("Generator State:\nxN={");
    for(i=0;i<12;i++) {printf("%llu",state->xN[i]%gq58x3_g); printf((i<11)?",":"}\nxP={");}
    for(i=0;i<12;i++) {printf("%llu",state->xP[i]%gq58x3_g); printf((i<11)?",":"}\n\n");}
}

__host__ void gq58x3_print_sse_state_(gq58x3_sse_state* state){int i;
    printf("Generator State:\nxN={");
    for(i=0;i<12;i++) {printf("%llu",state->xN[i]%gq58x3_g); printf((i<11)?",":"}\nxP={");}
    for(i=0;i<12;i++) {printf("%llu",state->xP[i]%gq58x3_g); printf((i<11)?",":"}\n\n");}
}

__global__ void gq58x3_kernel_generate_array(gq58x3_state* state, unsigned int* out, long* length) {
    unsigned sum,i,j,orbit,seqNum,shift1,shift2; long offset; lt temp;

    __shared__ lt xP[gq58x3_THREADS];        // one generator per s=12 threads, i.e. one orbit
    __shared__ lt xN[gq58x3_THREADS];        // per thread, i.e. blockDim.x orbits per block
    __shared__ unsigned  a[gq58x3_THREADS];  // array "a" contains corresponding parts of output

    orbit   = threadIdx.x % 12;
    seqNum  = (threadIdx.x + blockIdx.x * blockDim.x)/12;  // RNG_sequence index
    offset  = seqNum*(*length);                            // start of the section in the output array

    xP[threadIdx.x]=gq58x3_GetNextAny(state->xP[orbit],state->xN[orbit],0,offset);
    xN[threadIdx.x]=gq58x3_GetNextAny(state->xP[orbit],state->xN[orbit],0,offset+1);

    j=(orbit>>2); shift1 = 6-3*j; shift2 = (8*(orbit-(j<<2)));

    for(i=0;i<(*length);i++){       // each s=12 threads result in "length" values in the output array

      temp = gq58x3_CNext( xN[threadIdx.x], xP[threadIdx.x] );
      xP[threadIdx.x] = xN[threadIdx.x]; xN[threadIdx.x] = temp; 
      a[threadIdx.x]  = ((((temp/gq58x3_gdiv8)<<shift1)&(255U))<<shift2);

      __syncthreads();
      if((orbit&3)==0) a[threadIdx.x] = a[threadIdx.x]+a[threadIdx.x+1]+a[threadIdx.x+2]+a[threadIdx.x+3];
      __syncthreads();
      if(orbit==0){ sum=a[threadIdx.x]+a[threadIdx.x+4]+a[threadIdx.x+8];  out[offset+i]=sum; }

    }
}

__host__ void gq58x3_generate_gpu_array_(gq58x3_state* state, unsigned int* dev_out, long length){

   long          mylength = length/gq58x3_ARRAY_SECTIONS;
   gq58x3_state*   dev_state;
   long*         dev_length;

   if((mylength*gq58x3_ARRAY_SECTIONS)<length) mylength++;

   gq58x3_CUDA_CALL(hipMalloc((void**)&dev_state,sizeof(gq58x3_state)));
   gq58x3_CUDA_CALL(hipMalloc((void**)&dev_length,sizeof(long)));
   gq58x3_CUDA_CALL(hipMemcpy(dev_state,state,sizeof(gq58x3_state),hipMemcpyHostToDevice));
   gq58x3_CUDA_CALL(hipMemcpy(dev_length,&mylength,sizeof(long),hipMemcpyHostToDevice));

   gq58x3_kernel_generate_array<<<gq58x3_BLOCKS,gq58x3_THREADS>>>(dev_state,dev_out,dev_length);
   gq58x3_CUDA_CALL(hipGetLastError());
   
   gq58x3_CUDA_CALL(hipFree(dev_state)); gq58x3_CUDA_CALL(hipFree(dev_length));

}

__global__ void gq58x3_kernel_generate_array_float(gq58x3_state* state, float* out, long* length) {
    unsigned sum,i,j,orbit,seqNum,shift1,shift2; long offset; lt temp;

    __shared__ lt xP[gq58x3_THREADS];        // one generator per s=12 threads, i.e. one orbit
    __shared__ lt xN[gq58x3_THREADS];        // per thread, i.e. blockDim.x orbits per block
    __shared__ unsigned  a[gq58x3_THREADS];  // array "a" contains corresponding parts of output

    orbit   = threadIdx.x % 12;
    seqNum  = (threadIdx.x + blockIdx.x * blockDim.x)/12;  // RNG_sequence index
    offset  = seqNum*(*length);                            // start of the section in the output array

    xP[threadIdx.x]=gq58x3_GetNextAny(state->xP[orbit],state->xN[orbit],0,offset);
    xN[threadIdx.x]=gq58x3_GetNextAny(state->xP[orbit],state->xN[orbit],0,offset+1);

    j=(orbit>>2); shift1 = 6-3*j; shift2 = (8*(orbit-(j<<2)));

    for(i=0;i<(*length);i++){       // each s=12 threads result in "length" values in the output array

      temp = gq58x3_CNext( xN[threadIdx.x], xP[threadIdx.x] );
      xP[threadIdx.x] = xN[threadIdx.x]; xN[threadIdx.x] = temp; 
      a[threadIdx.x]  = ((((temp/gq58x3_gdiv8)<<shift1)&(255U))<<shift2);

      __syncthreads();
      if((orbit&3)==0) a[threadIdx.x] = a[threadIdx.x]+a[threadIdx.x+1]+a[threadIdx.x+2]+a[threadIdx.x+3];
      __syncthreads();
      if(orbit==0){ sum=a[threadIdx.x]+a[threadIdx.x+4]+a[threadIdx.x+8];  out[offset+i]=((float)sum) * 2.3283064365386963e-10; }

    }
}

__host__ void gq58x3_generate_gpu_array_float_(gq58x3_state* state, float* dev_out, long length){

   long          mylength = length/gq58x3_ARRAY_SECTIONS;
   gq58x3_state*   dev_state;
   long*         dev_length;

   if((mylength*gq58x3_ARRAY_SECTIONS)<length) mylength++;

   gq58x3_CUDA_CALL(hipMalloc((void**)&dev_state,sizeof(gq58x3_state)));
   gq58x3_CUDA_CALL(hipMalloc((void**)&dev_length,sizeof(long)));
   gq58x3_CUDA_CALL(hipMemcpy(dev_state,state,sizeof(gq58x3_state),hipMemcpyHostToDevice));
   gq58x3_CUDA_CALL(hipMemcpy(dev_length,&mylength,sizeof(long),hipMemcpyHostToDevice));

   gq58x3_kernel_generate_array_float<<<gq58x3_BLOCKS,gq58x3_THREADS>>>(dev_state,dev_out,dev_length);
   gq58x3_CUDA_CALL(hipGetLastError());
   
   gq58x3_CUDA_CALL(hipFree(dev_state)); gq58x3_CUDA_CALL(hipFree(dev_length));

}

__global__ void gq58x3_kernel_generate_array_double(gq58x3_state* state, double* out, long* length) {
    unsigned sum,i,j,orbit,seqNum,shift1,shift2; long offset; lt temp;

    __shared__ lt xP[gq58x3_THREADS];        // one generator per s=12 threads, i.e. one orbit
    __shared__ lt xN[gq58x3_THREADS];        // per thread, i.e. blockDim.x orbits per block
    __shared__ unsigned  a[gq58x3_THREADS];  // array "a" contains corresponding parts of output

    orbit   = threadIdx.x % 12;
    seqNum  = (threadIdx.x + blockIdx.x * blockDim.x)/12;  // RNG_sequence index
    offset  = seqNum*(*length);                            // start of the section in the output array

    xP[threadIdx.x]=gq58x3_GetNextAny(state->xP[orbit],state->xN[orbit],0,offset);
    xN[threadIdx.x]=gq58x3_GetNextAny(state->xP[orbit],state->xN[orbit],0,offset+1);

    j=(orbit>>2); shift1 = 6-3*j; shift2 = (8*(orbit-(j<<2)));

    for(i=0;i<(*length);i++){       // each s=12 threads result in "length" values in the output array

      temp = gq58x3_CNext( xN[threadIdx.x], xP[threadIdx.x] );
      xP[threadIdx.x] = xN[threadIdx.x]; xN[threadIdx.x] = temp; 
      a[threadIdx.x]  = ((((temp/gq58x3_gdiv8)<<shift1)&(255U))<<shift2);

      __syncthreads();
      if((orbit&3)==0) a[threadIdx.x] = a[threadIdx.x]+a[threadIdx.x+1]+a[threadIdx.x+2]+a[threadIdx.x+3];
      __syncthreads();
      if(orbit==0){ sum=a[threadIdx.x]+a[threadIdx.x+4]+a[threadIdx.x+8];  out[offset+i]=((double)sum) * 2.3283064365386963e-10; }

    }
}

__host__ void gq58x3_generate_gpu_array_double_(gq58x3_state* state, double* dev_out, long length){

   long          mylength = length/gq58x3_ARRAY_SECTIONS;
   gq58x3_state*   dev_state;
   long*         dev_length;

   if((mylength*gq58x3_ARRAY_SECTIONS)<length) mylength++;

   gq58x3_CUDA_CALL(hipMalloc((void**)&dev_state,sizeof(gq58x3_state)));
   gq58x3_CUDA_CALL(hipMalloc((void**)&dev_length,sizeof(long)));
   gq58x3_CUDA_CALL(hipMemcpy(dev_state,state,sizeof(gq58x3_state),hipMemcpyHostToDevice));
   gq58x3_CUDA_CALL(hipMemcpy(dev_length,&mylength,sizeof(long),hipMemcpyHostToDevice));

   gq58x3_kernel_generate_array_double<<<gq58x3_BLOCKS,gq58x3_THREADS>>>(dev_state,dev_out,dev_length);
   gq58x3_CUDA_CALL(hipGetLastError());
   
   gq58x3_CUDA_CALL(hipFree(dev_state)); gq58x3_CUDA_CALL(hipFree(dev_length));

}

__host__ void gq58x3_generate_array_(gq58x3_state* state, unsigned int* out, long length){

   long          mylength = length/gq58x3_ARRAY_SECTIONS;
   gq58x3_state*   dev_state;
   unsigned int* dev_out;
   long*         dev_length;

   if((mylength*gq58x3_ARRAY_SECTIONS)<length) mylength++;

   gq58x3_CUDA_CALL(hipMalloc((void**)&dev_state,sizeof(gq58x3_state)));
   gq58x3_CUDA_CALL(hipMalloc((void**)&dev_out,mylength*gq58x3_ARRAY_SECTIONS*sizeof(unsigned int)));
   gq58x3_CUDA_CALL(hipMalloc((void**)&dev_length,sizeof(long)));
   gq58x3_CUDA_CALL(hipMemcpy(dev_state,state,sizeof(gq58x3_state),hipMemcpyHostToDevice));
   gq58x3_CUDA_CALL(hipMemcpy(dev_length,&mylength,sizeof(long),hipMemcpyHostToDevice));

   gq58x3_kernel_generate_array<<<gq58x3_BLOCKS,gq58x3_THREADS>>>(dev_state,dev_out,dev_length);
   gq58x3_CUDA_CALL(hipGetLastError());
   
   gq58x3_CUDA_CALL(hipMemcpy(out,dev_out,length*sizeof(unsigned int),hipMemcpyDeviceToHost));
   gq58x3_CUDA_CALL(hipFree(dev_state)); gq58x3_CUDA_CALL(hipFree(dev_out)); 
   gq58x3_CUDA_CALL(hipFree(dev_length));

}
