// (c) Copyright 2013 Lev Barash, Landau Institute for Theoretical Physics, Russian Academy of Sciences
// This is supplement to the paper:
// L.Yu. Barash, L.N. Shchur, "PRAND: GPU accelerated parallel random number generation library: Using most reliable algorithms and applying parallelism of modern GPUs and CPUs".
// e-mail: barash @ itp.ac.ru (remove space)


#include <hip/hip_runtime.h>
#include<stdio.h>

#define gm29_CUDA_CALL(x) do { if((x) != hipSuccess) { printf("Error: %s at %s:%d\n",hipGetErrorString(hipGetLastError()),__FILE__,__LINE__); exit(1);}} while(0)

#define gm29_BLOCKS  512
#define gm29_THREADS 128
#define gm29_ARRAY_SECTIONS (gm29_BLOCKS*gm29_THREADS/32)

#define gm29_k 4
#define gm29_q 2
#define gm29_g 536870909U
#define gm29_halfg 268435456U

typedef struct{
  unsigned xN[32] __attribute__ ((aligned(16))), 
           xP[32] __attribute__ ((aligned(16)));
} gm29_state;

typedef gm29_state gm29_sse_state;

unsigned gm29_sse_Consts[16] __attribute__ ((aligned(16))) = 
{536870911,536870911,536870911,536870911,1073741818,1073741818,1073741818,1073741818,
 536870908,536870908,536870908,536870908,536870909,536870909,536870909,536870909};

__host__ unsigned int gm29_sse_generate_(gm29_sse_state* state){
  unsigned output1; unsigned output2 __attribute__ ((unused));
  asm volatile("movaps (%4),%%xmm7\n" \
      "movaps 16(%4),%%xmm6\n" \
      "movaps 32(%4),%%xmm4\n" \

      "movaps (%2),%%xmm0\n" \
      "movaps (%3),%%xmm5\n" \
      "movaps %%xmm0,(%3)\n" \
      "pslld  $2,%%xmm0\n" \
      "paddd  %%xmm6,%%xmm0\n" \
      "pslld  $1,%%xmm5\n" \
      "psubd  %%xmm5,%%xmm0\n" \
      "movaps %%xmm0,%%xmm5\n" \
      "psrld  $29,%%xmm5\n" \
      "pand   %%xmm7,%%xmm0\n" \
      "paddd  %%xmm5,%%xmm0\n" \
      "paddd  %%xmm5,%%xmm0\n" \
      "paddd  %%xmm5,%%xmm0\n" \
      "movaps %%xmm0,%%xmm5\n" \
      "pcmpgtd  %%xmm4,%%xmm5\n" \
      "pand   48(%4),%%xmm5\n" \
      "psubd  %%xmm5,%%xmm0\n" \
      "movaps %%xmm0,(%2)\n" \

      "movaps 16(%2),%%xmm1\n" \
      "movaps 16(%3),%%xmm5\n" \
      "movaps %%xmm1,16(%3)\n" \
      "pslld  $2,%%xmm1\n" \
      "paddd  %%xmm6,%%xmm1\n" \
      "pslld  $1,%%xmm5\n" \
      "psubd  %%xmm5,%%xmm1\n" \
      "movaps %%xmm1,%%xmm5\n" \
      "psrld  $29,%%xmm5\n" \
      "pand   %%xmm7,%%xmm1\n" \
      "paddd  %%xmm5,%%xmm1\n" \
      "paddd  %%xmm5,%%xmm1\n" \
      "paddd  %%xmm5,%%xmm1\n" \
      "movaps %%xmm1,%%xmm5\n" \
      "pcmpgtd  %%xmm4,%%xmm5\n" \
      "pand   48(%4),%%xmm5\n" \
      "psubd  %%xmm5,%%xmm1\n" \
      "movaps %%xmm1,16(%2)\n" \

      "movaps 32(%2),%%xmm2\n" \
      "movaps 32(%3),%%xmm5\n" \
      "movaps %%xmm2,32(%3)\n" \
      "pslld  $2,%%xmm2\n" \
      "paddd  %%xmm6,%%xmm2\n" \
      "pslld  $1,%%xmm5\n" \
      "psubd  %%xmm5,%%xmm2\n" \
      "movaps %%xmm2,%%xmm5\n" \
      "psrld  $29,%%xmm5\n" \
      "pand   %%xmm7,%%xmm2\n" \
      "paddd  %%xmm5,%%xmm2\n" \
      "paddd  %%xmm5,%%xmm2\n" \
      "paddd  %%xmm5,%%xmm2\n" \
      "movaps %%xmm2,%%xmm5\n" \
      "pcmpgtd  %%xmm4,%%xmm5\n" \
      "pand   48(%4),%%xmm5\n" \
      "psubd  %%xmm5,%%xmm2\n" \
      "movaps %%xmm2,32(%2)\n" \

      "movaps 48(%2),%%xmm3\n" \
      "movaps 48(%3),%%xmm5\n" \
      "movaps %%xmm3,48(%3)\n" \
      "pslld  $2,%%xmm3\n" \
      "paddd  %%xmm6,%%xmm3\n" \
      "pslld  $1,%%xmm5\n" \
      "psubd  %%xmm5,%%xmm3\n" \
      "movaps %%xmm3,%%xmm5\n" \
      "psrld  $29,%%xmm5\n" \
      "pand   %%xmm7,%%xmm3\n" \
      "paddd  %%xmm5,%%xmm3\n" \
      "paddd  %%xmm5,%%xmm3\n" \
      "paddd  %%xmm5,%%xmm3\n" \
      "movaps %%xmm3,%%xmm5\n" \
      "pcmpgtd  %%xmm4,%%xmm5\n" \
      "pand   48(%4),%%xmm5\n" \
      "psubd  %%xmm5,%%xmm3\n" \
      "movaps %%xmm3,48(%2)\n" \


      "psrld  $28,%%xmm0\n" \
      "psrld  $28,%%xmm1\n" \
      "psrld  $28,%%xmm2\n" \
      "psrld  $28,%%xmm3\n" \
      "packssdw %%xmm1,%%xmm0\n" \
      "packssdw %%xmm3,%%xmm2\n" \
      "packsswb %%xmm2,%%xmm0\n" \
      "psllw  $7,%%xmm0\n" \
      "pmovmskb %%xmm0,%0\n" \

      "movaps 64(%2),%%xmm0\n" \
      "movaps 64(%3),%%xmm5\n" \
      "movaps %%xmm0,64(%3)\n" \
      "pslld  $2,%%xmm0\n" \
      "paddd  %%xmm6,%%xmm0\n" \
      "pslld  $1,%%xmm5\n" \
      "psubd  %%xmm5,%%xmm0\n" \
      "movaps %%xmm0,%%xmm5\n" \
      "psrld  $29,%%xmm5\n" \
      "pand   %%xmm7,%%xmm0\n" \
      "paddd  %%xmm5,%%xmm0\n" \
      "paddd  %%xmm5,%%xmm0\n" \
      "paddd  %%xmm5,%%xmm0\n" \
      "movaps %%xmm0,%%xmm5\n" \
      "pcmpgtd  %%xmm4,%%xmm5\n" \
      "pand   48(%4),%%xmm5\n" \
      "psubd  %%xmm5,%%xmm0\n" \
      "movaps %%xmm0,64(%2)\n" \

      "movaps 80(%2),%%xmm1\n" \
      "movaps 80(%3),%%xmm5\n" \
      "movaps %%xmm1,80(%3)\n" \
      "pslld  $2,%%xmm1\n" \
      "paddd  %%xmm6,%%xmm1\n" \
      "pslld  $1,%%xmm5\n" \
      "psubd  %%xmm5,%%xmm1\n" \
      "movaps %%xmm1,%%xmm5\n" \
      "psrld  $29,%%xmm5\n" \
      "pand   %%xmm7,%%xmm1\n" \
      "paddd  %%xmm5,%%xmm1\n" \
      "paddd  %%xmm5,%%xmm1\n" \
      "paddd  %%xmm5,%%xmm1\n" \
      "movaps %%xmm1,%%xmm5\n" \
      "pcmpgtd  %%xmm4,%%xmm5\n" \
      "pand   48(%4),%%xmm5\n" \
      "psubd  %%xmm5,%%xmm1\n" \
      "movaps %%xmm1,80(%2)\n" \

      "movaps 96(%2),%%xmm2\n" \
      "movaps 96(%3),%%xmm5\n" \
      "movaps %%xmm2,96(%3)\n" \
      "pslld  $2,%%xmm2\n" \
      "paddd  %%xmm6,%%xmm2\n" \
      "pslld  $1,%%xmm5\n" \
      "psubd  %%xmm5,%%xmm2\n" \
      "movaps %%xmm2,%%xmm5\n" \
      "psrld  $29,%%xmm5\n" \
      "pand   %%xmm7,%%xmm2\n" \
      "paddd  %%xmm5,%%xmm2\n" \
      "paddd  %%xmm5,%%xmm2\n" \
      "paddd  %%xmm5,%%xmm2\n" \
      "movaps %%xmm2,%%xmm5\n" \
      "pcmpgtd  %%xmm4,%%xmm5\n" \
      "pand   48(%4),%%xmm5\n" \
      "psubd  %%xmm5,%%xmm2\n" \
      "movaps %%xmm2,96(%2)\n" \

      "movaps 112(%2),%%xmm3\n" \
      "movaps 112(%3),%%xmm5\n" \
      "movaps %%xmm3,112(%3)\n" \
      "pslld  $2,%%xmm3\n" \
      "paddd  %%xmm6,%%xmm3\n" \
      "pslld  $1,%%xmm5\n" \
      "psubd  %%xmm5,%%xmm3\n" \
      "movaps %%xmm3,%%xmm5\n" \
      "psrld  $29,%%xmm5\n" \
      "pand   %%xmm7,%%xmm3\n" \
      "paddd  %%xmm5,%%xmm3\n" \
      "paddd  %%xmm5,%%xmm3\n" \
      "paddd  %%xmm5,%%xmm3\n" \
      "movaps %%xmm3,%%xmm5\n" \
      "pcmpgtd  %%xmm4,%%xmm5\n" \
      "pand   48(%4),%%xmm5\n" \
      "psubd  %%xmm5,%%xmm3\n" \
      "movaps %%xmm3,112(%2)\n" \

      "psrld  $28,%%xmm0\n" \
      "psrld  $28,%%xmm1\n" \
      "psrld  $28,%%xmm2\n" \
      "psrld  $28,%%xmm3\n" \
      "packssdw %%xmm1,%%xmm0\n" \
      "packssdw %%xmm3,%%xmm2\n" \
      "packsswb %%xmm2,%%xmm0\n" \
      "psllw  $7,%%xmm0\n" \
      "pmovmskb %%xmm0,%1\n" \
      "shll $16,%1\n" \
      "addl %1,%0\n" \
      "":"=&r"(output1),"=&r"(output2):"r"(state->xN),"r"(state->xP),"r"(gm29_sse_Consts));
  return output1;
}

__device__ __host__ void gm29_get_sse_state_(gm29_state* state,gm29_sse_state* sse_state){
  int i; for(i=0;i<32;i++) {sse_state->xN[i]=state->xN[i]; sse_state->xP[i]=state->xP[i];}
}

__device__ __host__ unsigned gm29_CNext(unsigned N,unsigned P){
  return (gm29_k*N+gm29_q*(gm29_g-P))%gm29_g;
}

__device__ __host__ unsigned gm29_CNext2(unsigned N,unsigned P,unsigned myk,unsigned myq){
  unsigned long long NNN,PP,kk,qq,gg,rr;                    // returns (myk*N-myq*P) (mod gm29_g)
  NNN=N; PP=P; kk=myk; qq=myq; gg=gm29_g;
  rr=(kk*NNN+qq*(gg-PP));
  NNN=rr>>29;
  PP=rr-(NNN*gg);
  PP-=((PP>>29)*gg);
  return (unsigned)PP;
}

__device__ __host__ unsigned gm29_GetNextN(unsigned x0,unsigned x1,unsigned n){ //returns x_{2^n}
  unsigned myk=gm29_k,myq=gm29_q,i,x=x1;
  for(i=0;i<n;i++){
    x=gm29_CNext2(x,x0,myk,myq);
    myk=gm29_CNext2(myk,2,myk,myq);
    myq=gm29_CNext2(myq,0,myq,0);
  }
  return x;
}

__device__ __host__ unsigned gm29_GetNextAny(unsigned x0,unsigned x1,unsigned long long N){ // returns x_N
  unsigned long long i; unsigned xp=x0,xn=x1,xpnew,xnnew,shift=0;
  i=N; while(i>0){
    if(i%2==1){                        // xp,xn ----> 2^shift
      xpnew=gm29_GetNextN(xp,xn,shift);
      xnnew=gm29_GetNextN(xn,gm29_CNext(xn,xp),shift);
      xp=xpnew; xn=xnnew;
    }
    i/=2; shift++;
  }
  return xp;
}

__device__ __host__ void gm29_skipahead_(gm29_state* state, unsigned long long offset){
  unsigned xn,xp,j; 
  for(j=0;j<32;j++){
    xp=gm29_GetNextAny(state->xP[j],state->xN[j],offset);
    xn=gm29_GetNextAny(state->xP[j],state->xN[j],offset+1);
    state->xP[j]=xp; state->xN[j]=xn;
  }
}

__device__ __host__ void gm29_init_(gm29_state* state){
   unsigned x0=514932,x1=127293,xp,xn,j;
   for(j=0;j<32;j++){
     xp=gm29_GetNextAny(x0,x1,9007198285571818UL);
     xn=gm29_GetNextAny(x0,x1,9007198285571819UL);
     state->xP[j]=xp; state->xN[j]=xn; x0=xp; x1=xn;
   }
}

__device__ __host__ void gm29_init_short_sequence_(gm29_state* state,unsigned SequenceNumber){
  gm29_init_(state);                     // 0 <= SequenceNumber < 10^8;   length of each sequence <= 8*10^7
  gm29_skipahead_(state,82927047ULL*(unsigned long long)SequenceNumber);
}

__device__ __host__ void gm29_init_medium_sequence_(gm29_state* state,unsigned SequenceNumber){
  gm29_init_(state);                     // 0 <= SequenceNumber < 10^6;   length of each sequence <= 8*10^9
  gm29_skipahead_(state,8799201913ULL*(unsigned long long)SequenceNumber);
}

__device__ __host__ void gm29_init_long_sequence_(gm29_state* state,unsigned SequenceNumber){
  gm29_init_(state);                     // 0 <= SequenceNumber < 10^4;   length of each sequence <= 8*10^11
  gm29_skipahead_(state,828317697521ULL*(unsigned long long)SequenceNumber);
}

__device__ __host__ unsigned int gm29_generate_(gm29_state* state){
  unsigned sum=0, i, temp, bit=1;
  for(i=0;i<32;i++){ 
    temp=(gm29_k*state->xN[i]+gm29_q*(gm29_g-state->xP[i]))%gm29_g;
    state->xP[i]=state->xN[i]; state->xN[i]=temp;
    sum+= ((temp<gm29_halfg)?0:bit); bit*=2;
  }
  return sum;
}

__device__ __host__ float gm29_generate_uniform_float_(gm29_state* state){
  unsigned sum=0, i, temp,bit=1;
  for(i=0;i<32;i++){ 
    temp=(gm29_k*state->xN[i]+gm29_q*(gm29_g-state->xP[i]))%gm29_g;
    state->xP[i]=state->xN[i]; state->xN[i]=temp;
    sum+= ((temp<gm29_halfg)?0:bit); bit*=2;
  }
  return ((float) sum) * 2.3283064365386963e-10;
}

__host__ void gm29_print_state_(gm29_state* state){int i;
    printf("Generator State:\nxN={");
    for(i=0;i<32;i++) {printf("%u",state->xN[i]%gm29_g); printf((i<31)?",":"}\nxP={");}
    for(i=0;i<32;i++) {printf("%u",state->xP[i]%gm29_g); printf((i<31)?",":"}\n\n");}
}

__host__ void gm29_print_sse_state_(gm29_sse_state* state){int i;
    printf("Generator State:\nxN={");
    for(i=0;i<32;i++) {printf("%u",state->xN[i]%gm29_g); printf((i<31)?",":"}\nxP={");}
    for(i=0;i<32;i++) {printf("%u",state->xP[i]%gm29_g); printf((i<31)?",":"}\n\n");}
}

__global__ void gm29_kernel_generate_array(gm29_state* state, unsigned int* out, long* length) {
    unsigned temp,sum,i,orbit,seqNum; long offset;

    __shared__ unsigned xP[gm29_THREADS];  // one generator per s=32 threads, i.e. one orbit
    __shared__ unsigned xN[gm29_THREADS];  // per thread, i.e. blockDim.x orbits per block
    __shared__ unsigned  a[gm29_THREADS];  // array "a" contains corresponding parts of output

    orbit   = threadIdx.x % 32;
    seqNum  = (threadIdx.x + blockIdx.x * blockDim.x)>>5;  // RNG_sequence index
    offset  = seqNum*(*length);                            // start of the section in the output array

    xP[threadIdx.x]=gm29_GetNextAny(state->xP[orbit],state->xN[orbit],offset);
    xN[threadIdx.x]=gm29_GetNextAny(state->xP[orbit],state->xN[orbit],offset+1);

    for(i=0;i<(*length);i++){

      temp = gm29_CNext( xN[threadIdx.x], xP[threadIdx.x] );
      xP[threadIdx.x] = xN[threadIdx.x]; xN[threadIdx.x] = temp; 
      a[threadIdx.x]  = (temp < gm29_halfg ? 0 : (1<<orbit) );

      __syncthreads();              // each s=32 threads result in "length" values in the output array
      if((orbit&3)==0) a[threadIdx.x] = a[threadIdx.x]+a[threadIdx.x+1]+a[threadIdx.x+2]+a[threadIdx.x+3];
      __syncthreads();
      if((orbit&15)==0) a[threadIdx.x] = a[threadIdx.x]+a[threadIdx.x+4]+a[threadIdx.x+8]+a[threadIdx.x+12];
      __syncthreads();
      if(orbit==0){ sum=a[threadIdx.x]+a[threadIdx.x+16];  out[offset+i]=sum; }

    }
}

__host__ void gm29_generate_gpu_array_(gm29_state* state, unsigned int* dev_out, long length){

   long          mylength = length/gm29_ARRAY_SECTIONS;
   gm29_state*   dev_state;
   long*         dev_length;

   if((mylength*gm29_ARRAY_SECTIONS)<length) mylength++;

   gm29_CUDA_CALL(hipMalloc((void**)&dev_state,sizeof(gm29_state)));
   gm29_CUDA_CALL(hipMalloc((void**)&dev_length,sizeof(long)));
   gm29_CUDA_CALL(hipMemcpy(dev_state,state,sizeof(gm29_state),hipMemcpyHostToDevice));
   gm29_CUDA_CALL(hipMemcpy(dev_length,&mylength,sizeof(long),hipMemcpyHostToDevice));

   gm29_kernel_generate_array<<<gm29_BLOCKS,gm29_THREADS>>>(dev_state,dev_out,dev_length);
   gm29_CUDA_CALL(hipGetLastError());
   
   gm29_CUDA_CALL(hipFree(dev_state)); gm29_CUDA_CALL(hipFree(dev_length));

}

__global__ void gm29_kernel_generate_array_float(gm29_state* state, float* out, long* length) {
    unsigned temp,sum,i,orbit,seqNum; long offset;

    __shared__ unsigned xP[gm29_THREADS];  // one generator per s=32 threads, i.e. one orbit
    __shared__ unsigned xN[gm29_THREADS];  // per thread, i.e. blockDim.x orbits per block
    __shared__ unsigned  a[gm29_THREADS];  // array "a" contains corresponding parts of output

    orbit   = threadIdx.x % 32;
    seqNum  = (threadIdx.x + blockIdx.x * blockDim.x)>>5;  // RNG_sequence index
    offset  = seqNum*(*length);                            // start of the section in the output array

    xP[threadIdx.x]=gm29_GetNextAny(state->xP[orbit],state->xN[orbit],offset);
    xN[threadIdx.x]=gm29_GetNextAny(state->xP[orbit],state->xN[orbit],offset+1);

    for(i=0;i<(*length);i++){      // each s=32 threads result in "length" values in the output array

      temp = gm29_CNext( xN[threadIdx.x], xP[threadIdx.x] );
      xP[threadIdx.x] = xN[threadIdx.x]; xN[threadIdx.x] = temp; 
      a[threadIdx.x]  = (temp < gm29_halfg ? 0 : (1<<orbit) );

      __syncthreads();
      if((orbit&3)==0) a[threadIdx.x] = a[threadIdx.x]+a[threadIdx.x+1]+a[threadIdx.x+2]+a[threadIdx.x+3];
      __syncthreads();
      if((orbit&15)==0) a[threadIdx.x] = a[threadIdx.x]+a[threadIdx.x+4]+a[threadIdx.x+8]+a[threadIdx.x+12];
      __syncthreads();
      if(orbit==0){ sum=a[threadIdx.x]+a[threadIdx.x+16];  out[offset+i]=((float)sum) * 2.3283064365386963e-10; }

    }
}

__host__ void gm29_generate_gpu_array_float_(gm29_state* state, float* dev_out, long length){

   long          mylength = length/gm29_ARRAY_SECTIONS;
   gm29_state*   dev_state;
   long*         dev_length;

   if((mylength*gm29_ARRAY_SECTIONS)<length) mylength++;

   gm29_CUDA_CALL(hipMalloc((void**)&dev_state,sizeof(gm29_state)));
   gm29_CUDA_CALL(hipMalloc((void**)&dev_length,sizeof(long)));
   gm29_CUDA_CALL(hipMemcpy(dev_state,state,sizeof(gm29_state),hipMemcpyHostToDevice));
   gm29_CUDA_CALL(hipMemcpy(dev_length,&mylength,sizeof(long),hipMemcpyHostToDevice));

   gm29_kernel_generate_array_float<<<gm29_BLOCKS,gm29_THREADS>>>(dev_state,dev_out,dev_length);
   gm29_CUDA_CALL(hipGetLastError());
   
   gm29_CUDA_CALL(hipFree(dev_state)); gm29_CUDA_CALL(hipFree(dev_length));

}

__global__ void gm29_kernel_generate_array_double(gm29_state* state, double* out, long* length) {
    unsigned temp,sum,i,orbit,seqNum; long offset;

    __shared__ unsigned xP[gm29_THREADS];  // one generator per s=32 threads, i.e. one orbit
    __shared__ unsigned xN[gm29_THREADS];  // per thread, i.e. blockDim.x orbits per block
    __shared__ unsigned  a[gm29_THREADS];  // array "a" contains corresponding parts of output

    orbit   = threadIdx.x % 32;
    seqNum  = (threadIdx.x + blockIdx.x * blockDim.x)>>5;  // RNG_sequence index
    offset  = seqNum*(*length);                            // start of the section in the output array

    xP[threadIdx.x]=gm29_GetNextAny(state->xP[orbit],state->xN[orbit],offset);
    xN[threadIdx.x]=gm29_GetNextAny(state->xP[orbit],state->xN[orbit],offset+1);

    for(i=0;i<(*length);i++){       // each s=32 threads result in "length" values in the output array

      temp = gm29_CNext( xN[threadIdx.x], xP[threadIdx.x] );
      xP[threadIdx.x] = xN[threadIdx.x]; xN[threadIdx.x] = temp; 
      a[threadIdx.x]  = (temp < gm29_halfg ? 0 : (1<<orbit) );

      __syncthreads();
      if((orbit&3)==0) a[threadIdx.x] = a[threadIdx.x]+a[threadIdx.x+1]+a[threadIdx.x+2]+a[threadIdx.x+3];
      __syncthreads();
      if((orbit&15)==0) a[threadIdx.x] = a[threadIdx.x]+a[threadIdx.x+4]+a[threadIdx.x+8]+a[threadIdx.x+12];
      __syncthreads();
      if(orbit==0){ sum=a[threadIdx.x]+a[threadIdx.x+16];  out[offset+i]=((double)sum) * 2.3283064365386963e-10; }

    }
}

__host__ void gm29_generate_gpu_array_double_(gm29_state* state, double* dev_out, long length){

   long          mylength = length/gm29_ARRAY_SECTIONS;
   gm29_state*   dev_state;
   long*         dev_length;

   if((mylength*gm29_ARRAY_SECTIONS)<length) mylength++;

   gm29_CUDA_CALL(hipMalloc((void**)&dev_state,sizeof(gm29_state)));
   gm29_CUDA_CALL(hipMalloc((void**)&dev_length,sizeof(long)));
   gm29_CUDA_CALL(hipMemcpy(dev_state,state,sizeof(gm29_state),hipMemcpyHostToDevice));
   gm29_CUDA_CALL(hipMemcpy(dev_length,&mylength,sizeof(long),hipMemcpyHostToDevice));

   gm29_kernel_generate_array_double<<<gm29_BLOCKS,gm29_THREADS>>>(dev_state,dev_out,dev_length);
   gm29_CUDA_CALL(hipGetLastError());
   
   gm29_CUDA_CALL(hipFree(dev_state)); gm29_CUDA_CALL(hipFree(dev_length));

}

__host__ void gm29_generate_array_(gm29_state* state, unsigned int* out, long length){

   long          mylength = length/gm29_ARRAY_SECTIONS;
   gm29_state*   dev_state;
   unsigned int* dev_out;
   long*         dev_length;

   if((mylength*gm29_ARRAY_SECTIONS)<length) mylength++;

   gm29_CUDA_CALL(hipMalloc((void**)&dev_state,sizeof(gm29_state)));
   gm29_CUDA_CALL(hipMalloc((void**)&dev_out,mylength*gm29_ARRAY_SECTIONS*sizeof(unsigned int)));
   gm29_CUDA_CALL(hipMalloc((void**)&dev_length,sizeof(long)));
   gm29_CUDA_CALL(hipMemcpy(dev_state,state,sizeof(gm29_state),hipMemcpyHostToDevice));
   gm29_CUDA_CALL(hipMemcpy(dev_length,&mylength,sizeof(long),hipMemcpyHostToDevice));

   gm29_kernel_generate_array<<<gm29_BLOCKS,gm29_THREADS>>>(dev_state,dev_out,dev_length);
   gm29_CUDA_CALL(hipGetLastError());
   
   gm29_CUDA_CALL(hipMemcpy(out,dev_out,length*sizeof(unsigned int),hipMemcpyDeviceToHost));
   gm29_CUDA_CALL(hipFree(dev_state)); gm29_CUDA_CALL(hipFree(dev_out)); 
   gm29_CUDA_CALL(hipFree(dev_length));

}
