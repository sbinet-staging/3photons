
#include<stdio.h>
#include<gm31.h>

#define NN       100000000UL

int main(void){ 
   long i; unsigned int sum=0;
   gm31_state state;
   gm31_init_(&state);
   gm31_print_state_(&state);
   for(i=0;i<NN;i++) sum+=gm31_generate_(&state);
   printf("%ld GM31 pseudorandom numbers generated using general instructions of CPU.\n",NN);
   printf("Fractional part of the total sum of generated numbers: %f\n",sum/4294967296.);
   printf("Next output value: %f\n",gm31_generate_(&state)/4294967296.);
   return 0;
}
